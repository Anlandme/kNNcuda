
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <math.h>
#include <float.h>


struct betterKdTree {
	//node and leef
	float value;
	struct betterKdTree *left, *right;
};

struct Matrix {
    int width;
    int height;
    int stride; 
    float* elements;
};

struct MatrixInt {
	int width;
    int height;
    int stride; 
    int* elements;
};

void mallocMatrix(Matrix& m){
	m.elements = (float*)malloc(m.width * m.height * sizeof(float));
}


void generateHostMatrix(Matrix& m, int width, int height){
	
	m.height = height;
	m.width = width;
	m.stride = m.width;
	mallocMatrix(m);
	
}

void fillMatrixWithRandomFloats(Matrix& m)
{
	int i;
	//int j;
	for(i=0;i<m.height*m.width;i++)
	{
		*(m.elements + i ) = (((float)rand()/(float)(RAND_MAX)) * 10.0 - 5.0);
	}
}

void fillMatrixWithSequence(Matrix& m){
	for(int i=0;i<m.width*m.height;i++){
		m.elements[i] = i;
	}
}  

void copyRowToMatrix(Matrix& in, int row, Matrix& out){
	for(int i = 0; i<in.width*(row+1) && i<out.width; i++){
		out.elements[i] = in.elements[i+in.width*(row)];
	}
}

void copyVectorInterval(Matrix& in,int start, int end, Matrix& out){
	for(int i=0; i < (end-start); i++){
		out.elements[i] = in.elements[i+start];
	}
}

void copyVectorValuesUnderOverLimit(Matrix& V, MatrixInt in, float limit, int median, int dim, MatrixInt& outUnder, MatrixInt& outOver){
	int i_l = 0;
	int i_r = 0;
	for(int i=0; i < in.width; i++){
		
		if(in.elements[i] > V.width){
			printf("in.elements[%d] = %d\n",i,in.elements[i]);
		}
		
		
		if( V.elements[ dim*V.width + in.elements[i] ] <= limit && i_l < outUnder.width ){
			//~ printf("V[dim:%d][%d] : %f < %f\n",dim,in.elements[i],V.elements[ dim*V.width + in.elements[i] ],limit);
			outUnder.elements[i_l++] = in.elements[i];
			
		}else{
			outOver.elements[i_r++] = in.elements[i];
		}
		
	}
	
	if(i_r < outOver.width || i_l < outUnder.width){
		printf("AAAR total_indices: %d, median: %d, split: %f\n", in.width, median, limit);
		for(int i=0; i < in.width; i++){
			printf("in.elements[%d] = %d, value: %f\n", i, in.elements[i], V.elements[dim*V.width +in.elements[i] ]);
		}
		
		for(int i=0; i< outUnder.width; i++){
			printf("outUnder.elements[%d] = %d\n", i, outUnder.elements[i]);
		}
		
		for(int i=0; i< outOver.width; i++){
			printf("outOver.elements[%d] = %d\n", i, outOver.elements[i]);
		}
		
		throw 20;
	}
}



void mergeHostWithIndices(float* a, float* b, int i1, int j1, int i2, int j2,int limit=-1){
	int limit_end = limit;
	
	
	float* temp = (float*) malloc((j2-i1+1) * sizeof(float));  //array used for merging
    int* temp_indices = (int*) malloc((j2-i1+1) * sizeof(int));  //array used for merging
    
    
    int i,j,k;
    i=i1;    //beginning of the first list
    j=i2;    //beginning of the second list
    k=0;
    
    int counter = 0;
    while(i<=j1 && j<=j2 && limit!=0)    //while elements in both lists
    {
		counter ++;
		limit--;
        if(a[i]<a[j]){
			temp_indices[k] = b[i]; 
            temp[k++]=a[i++];
            
        }else{
			temp_indices[k] = b[j];
            temp[k++]=a[j++];
		}
    }
    
    while(i<=j1 && limit!=0) {   //copy remaining elements of the first list
		temp_indices[k] = b[i]; 
        temp[k++]=a[i++];
	}
        
    while(j<=j2 && limit!=0) {   //copy remaining elements of the second list
        temp_indices[k] = b[j]; 
        temp[k++]=a[j++];
	}
        
    //Transfer elements from temp[] back to a[]
    for(i=i1,j=0;i<=j2 && limit_end!=0 ;i++,j++,limit_end--)
	{
		b[i] = temp_indices[j];
		if(b[i] < 0){
			printf("THERE IS SOMETHING WRONG\n");
		}
        a[i] = temp[j];
    }   
    free(temp_indices);
    free(temp);
}


void naturalMergeSort(Matrix& in, int dim, Matrix& indices,  Matrix& m, int limit=-1){
	
	copyRowToMatrix(in, dim, m);
	
	//~ printf("copy row of Mat for dim %d\n", dim);
	//~ printMatrix(m,true);
	
	int m_elements = m.width * m.height;
	
	int slide_buffer_size = int(m_elements-0.5);
	int* slide_buffer = (int*) malloc(slide_buffer_size * sizeof(int));

	clock_t calcstart, calcend;
	calcstart = clock();

	//create RUNS
	int num_slides = 1;
	slide_buffer[0] = 0;
	for(int i=1; i < slide_buffer_size+1; i++) {
		if(m.elements[i] < m.elements[i-1])
		{
			slide_buffer[num_slides] = i;
			num_slides++;
		}
		
	}
	slide_buffer[num_slides] = m_elements;
	slide_buffer_size = num_slides+1;
	
	
	//sort 
	int count = 0;
	int current_limit = -1;
	while(num_slides > 1){
		if(num_slides > 2){
			current_limit = limit;
		}
		//~ std::cout << count+1 <<" Iteration: You can use " << int(num_slides/2) << " Threads" << std::endl;
		
		int i;
		
		for(i=2;i<int(num_slides+1);i+=2)
		{
				
			mergeHostWithIndices(m.elements, indices.elements ,slide_buffer[i-2], slide_buffer[i-1]-1, slide_buffer[i-1], slide_buffer[i]-1,current_limit);
			
			
			slide_buffer[i/2]= slide_buffer[i];
		}
		
		if(num_slides%2 == 1){
			slide_buffer[(num_slides+1)/2] = slide_buffer[num_slides];
		}
		
		count ++;
		num_slides = int(num_slides/2.0+0.5);
		
	}
	
	calcend = clock();
	printf("Sort CPU %f milliseconds\n",(float)(calcend-calcstart)*1000.0 / CLOCKS_PER_SEC);
	
	free(slide_buffer);
}

void sortByDim(Matrix& V, int dim, Matrix& indices, Matrix& values){
	naturalMergeSort(V, dim, indices, values);
}

void calculateMedian(Matrix& V, Matrix& indices, int current_dim, float& median_index, float& median_value) {
	median_index = indices.elements[indices.width/2];
	median_value = V.elements[V.width*current_dim+(int)median_index];
}

void generateKdTreeArray(Matrix& V, Matrix* sorted_indices, int max_dim, Matrix& kd_tree, int& size, int& max_tree_depth){
	printf("START\n");
	max_tree_depth = static_cast<int>(log2f(V.width-1)+2.0) ;
	if(V.width == 1){
		max_tree_depth = 1;
	}
	printf("tree depth: %d\n",max_tree_depth);
	
	size = pow(2, max_tree_depth) - 1;
	
	printf("calulated kd-tree size: %d\n",size);
	generateHostMatrix(kd_tree, size, 1);
	
	
	float median_index, median_value;
	int current_dim = 0;
	for(int i = 0; i<max_tree_depth; i++){
		current_dim = i%max_dim;
		calculateMedian(V, sorted_indices[current_dim], current_dim, median_index, median_value);
		printf("dim: %d: Median Index: %f, Value: %f\n", current_dim, median_index, median_value);
	}
	
}


int main(int argc, char** argv)
{
	printf("Hello World!\n");
	
	const char * filename  = "points.ply";
	
	//HOST STUFF
	int point_dim = 3;
	int num_points = 4;
	
	if(argc > 1){
		num_points = atoi(argv[1]);
	}
	
	int k=50;
	int dim_points = 3;
	
	Matrix V;
	struct Matrix test;
	struct Matrix indices_sorted[point_dim];
	struct Matrix values_sorted[point_dim];
	
	generateHostMatrix(test, num_points, 1);
	generateHostMatrix( V, num_points, point_dim);
	fillMatrixWithRandomFloats( V);
	
	for(int i=0; i < point_dim; i++)
	{
		printf("generate indices for dim %d\n",i);
		generateHostMatrix(indices_sorted[i], V.width, 1);
		
		printf("generate values for dim %d\n",i);
		generateHostMatrix(values_sorted[i], V.width,1);
		fillMatrixWithSequence(indices_sorted[i]);
		
		sortByDim( V, i, indices_sorted[i] , values_sorted[i]);
	}
	
	
	printf("Start generating kd-tree array based\n");
	//sorted indices + values
	//do some stuff
	Matrix kd_tree;
	int size = 0;
	int max_tree_depth = 0;
	
	generateKdTreeArray(V, indices_sorted, dim_points, kd_tree, size, max_tree_depth);
	
	printf("End generating kd-tree array based\n");
	
	printf("Free kd_tree array\n");
	free(kd_tree.elements);
	
	for(int i=0; i<point_dim;i++)
	{
		printf("free indices dim %d\n",i+1);
		free(indices_sorted[i].elements);
		
		printf("free values dim %d\n",i+1);
		free(values_sorted[i].elements);
	}
	
	free(V.elements);
	free(test.elements);
}
