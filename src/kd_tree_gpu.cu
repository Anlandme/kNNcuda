#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <math.h>
#include <float.h>
#include "../include/hip/hip_runtime_api.h"

int m_mps = 0;
int m_cuda_cores_per_mp = 0;
int m_threads_per_mp = 0;
int m_threads_per_block = 0;
int* m_size_thread_block = new int(3);
int* m_size_grid = new int(3);
unsigned long long m_device_global_memory = 0;

struct betterKdTree {
	//node and leef
	float value;
	struct betterKdTree *left, *right;
};

struct Matrix {
    int width;
    int height;
    int stride; 
    float* elements;
};

struct MatrixInt {
	int width;
    int height;
    int stride; 
    int* elements;
};

void printMatrix(Matrix& m)
{
	int i;
	//int j;
	for(i=0;i<m.width*m.height;i++)
	{
		if(i%m.width == 0){
			printf("|");
		}
		printf(" %f ",*(m.elements + i ));
		if(i%m.width == m.width-1){
			printf("|\n");
		}
	}
	
	printf("\n");
}




void mallocMatrix(Matrix& m){
	m.elements = (float*)malloc(m.width * m.height * sizeof(float));
}


void generateHostMatrix(Matrix& m, int width, int height){
	
	
	m.height = height;
	m.width = width;
	m.stride = m.width;
	m.elements = (float*)malloc(m.width * m.height * sizeof(float) );
	
}

void generateDeviceMatrix(Matrix& D_m, int width, int height){
	
    D_m.width = D_m.stride = width;
    D_m.height = height;
    size_t size = D_m.width * D_m.height * sizeof(float);
    hipMalloc(&D_m.elements, size);
    
}

void copyToDeviceMatrix(Matrix& m, Matrix& D_m){
	
	size_t size = m.width * m.height * sizeof(float);
    hipMemcpy(D_m.elements, m.elements, size, hipMemcpyHostToDevice);

}

void copyToHostMatrix(Matrix& D_m, Matrix& m){
	
	size_t size = m.width * m.height * sizeof(float);
	printf("size: %d\n", (int)size);
	hipMemcpy(m.elements, D_m.elements, size, hipMemcpyDeviceToHost);
	
}

void fillMatrixWithRandomFloats(Matrix& m)
{
	int i;
	//int j;
	for(i=0;i<m.height*m.width;i++)
	{
		*(m.elements + i ) = (((float)rand()/(float)(RAND_MAX)) * 10.0 - 5.0);
	}
}

void fillMatrixWithSequence(Matrix& m){
	for(int i=0;i<m.width*m.height;i++){
		m.elements[i] = i;
	}
}  

void copyRowToMatrix(Matrix& in, int row, Matrix& out){
	for(int i = 0; i<in.width*(row+1) && i<out.width; i++){
		out.elements[i] = in.elements[i+in.width*(row)];
	}
}

void copyVectorInterval(Matrix& in,int start, int end, Matrix& out){
	for(int i=0; i < (end-start); i++){
		out.elements[i] = in.elements[i+start];
	}
}




__global__ void KNNKernel(const Matrix D_V, const Matrix D_kd_tree, Matrix D_Result_Normals, int k=50);

__global__ void FlipNormalsKernel(const Matrix D_V, Matrix D_Result_Normals, float x = 1000.0, float y=1000.0, float z=1000.0);



void getCudaInformation(int& mps, int& cuda_cores_per_mp, int& threads_per_mp, int& threads_per_block, int* size_thread_block, int* size_grid , unsigned long long& device_global_memory){
	hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    
    
    mps = deviceProp.multiProcessorCount;
    cuda_cores_per_mp = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
    threads_per_mp = deviceProp.maxThreadsPerMultiProcessor;
    threads_per_block = deviceProp.maxThreadsPerBlock;
    size_thread_block[0] = deviceProp.maxThreadsDim[0];
    size_thread_block[1] = deviceProp.maxThreadsDim[1];
    size_thread_block[2] = deviceProp.maxThreadsDim[2];
    size_grid[0] = deviceProp.maxGridSize[0];
    size_grid[1] = deviceProp.maxGridSize[1];
    size_grid[2] = deviceProp.maxGridSize[2];
    device_global_memory = (unsigned long long) deviceProp.totalGlobalMem;
    
}


void mergeHostWithIndices(float* a, float* b, int i1, int j1, int i2, int j2,int limit=-1){
	int limit_end = limit;
	
	
	float* temp = (float*) malloc((j2-i1+1) * sizeof(float));  //array used for merging
    int* temp_indices = (int*) malloc((j2-i1+1) * sizeof(int));  //array used for merging
    
    
    int i,j,k;
    i=i1;    //beginning of the first list
    j=i2;    //beginning of the second list
    k=0;
    
    int counter = 0;
    while(i<=j1 && j<=j2 && limit!=0)    //while elements in both lists
    {
		counter ++;
		limit--;
        if(a[i]<a[j]){
			temp_indices[k] = b[i]; 
            temp[k++]=a[i++];
            
        }else{
			temp_indices[k] = b[j];
            temp[k++]=a[j++];
		}
    }
    
    while(i<=j1 && limit!=0) {   //copy remaining elements of the first list
		temp_indices[k] = b[i]; 
        temp[k++]=a[i++];
	}
        
    while(j<=j2 && limit!=0) {   //copy remaining elements of the second list
        temp_indices[k] = b[j]; 
        temp[k++]=a[j++];
	}
        
    //Transfer elements from temp[] back to a[]
    for(i=i1,j=0;i<=j2 && limit_end!=0 ;i++,j++,limit_end--)
	{
		b[i] = temp_indices[j];
		if(b[i] < 0){
			printf("THERE IS SOMETHING WRONG\n");
		}
        a[i] = temp[j];
    }   
    free(temp_indices);
    free(temp);
}


void naturalMergeSort(Matrix& in, int dim, Matrix& indices,  Matrix& m, int limit=-1){
	
	copyRowToMatrix(in, dim, m);
	
	//~ printf("copy row of Mat for dim %d\n", dim);
	//~ printMatrix(m,true);
	
	int m_elements = m.width * m.height;
	
	int slide_buffer_size = int(m_elements-0.5);
	int* slide_buffer = (int*) malloc(slide_buffer_size * sizeof(int));

	clock_t calcstart, calcend;
	calcstart = clock();

	//create RUNS
	int num_slides = 1;
	slide_buffer[0] = 0;
	for(int i=1; i < slide_buffer_size+1; i++) {
		if(m.elements[i] < m.elements[i-1])
		{
			slide_buffer[num_slides] = i;
			num_slides++;
		}
		
	}
	slide_buffer[num_slides] = m_elements;
	slide_buffer_size = num_slides+1;
	
	
	//sort 
	int count = 0;
	int current_limit = -1;
	while(num_slides > 1){
		if(num_slides > 2){
			current_limit = limit;
		}
		//~ std::cout << count+1 <<" Iteration: You can use " << int(num_slides/2) << " Threads" << std::endl;
		
		int i;
		
		for(i=2;i<int(num_slides+1);i+=2)
		{
				
			mergeHostWithIndices(m.elements, indices.elements ,slide_buffer[i-2], slide_buffer[i-1]-1, slide_buffer[i-1], slide_buffer[i]-1,current_limit);
			
			
			slide_buffer[i/2]= slide_buffer[i];
		}
		
		if(num_slides%2 == 1){
			slide_buffer[(num_slides+1)/2] = slide_buffer[num_slides];
		}
		
		count ++;
		num_slides = int(num_slides/2.0+0.5);
		
	}
	
	calcend = clock();
	printf("Sort CPU %f milliseconds\n",(float)(calcend-calcstart)*1000.0 / CLOCKS_PER_SEC);
	
	free(slide_buffer);
}

void sortByDim(Matrix& V, int dim, Matrix& indices, Matrix& values){
	naturalMergeSort(V, dim, indices, values);
}

void calculateMedian(Matrix& V, Matrix& indices, int current_dim, float& median_index, float& median_value, int num_medians) {
	
	median_index = indices.elements[indices.width/2+1];
	median_value = V.elements[V.width*current_dim+(int)median_index];
}



void splitMatrix(Matrix& I, Matrix& I_L, Matrix& I_R){
	
	int i=0;
	for(; i<I_L.width; i++){
		I_L.elements[i] = I.elements[i];
	}
	int j=0;
	for(; i<I.width && j<I_R.width; i++, j++){
		I_R.elements[j] = I.elements[i];
	}
	
}

void splitMatrixWithValue(Matrix& V, Matrix& I, Matrix& I_L, Matrix& I_R, int current_dim, float value){
	int i_l = 0;
	int i_r = 0;
	
	//~ printMatrix(V);
	//~ printMatrix(I);
	
	//~ printf("split by value: %f\n",value);
	//~ printf("splitting array (%d) to (%d, %d) with value %f\n",I.width,I_L.width,I_R.width,value);
	for(int i=0; i<I.width; i++){
		float current_value = V.elements[current_dim*V.width + static_cast<int>(I.elements[i]+0.5) ];
		//~ printf("curr val: %f\n", current_value);
		if(current_value <= value && I_L.width > i_l ){
			//~ printf("add to left: %f with value %f\n", I.elements[i], current_value);
			I_L.elements[i_l++] = I.elements[i];
		}else if(current_value >= value && I_R.width > i_r){
			//~ printf("add to right: %f with value %f\n", I.elements[i], current_value);
			I_R.elements[i_r++] = I.elements[i];
		}else {
			if(i_r<I_R.width){
				I_R.elements[i_r++] = I.elements[i];
			}else if(i_l<I_L.width){
				I_L.elements[i_l++] = I.elements[i];
			}
		}
	}
	
	if(i_l != I_L.width){
		printf("WARNING left %d != %d\n",i_l,I_L.width);
	}
	
	if(i_r != I_R.width){
		printf("WARNING right %d != %d\n",i_r,I_R.width);
	}
		
}

void generateKdTreeRecursive(Matrix& V, Matrix* sorted_indices, int current_dim, int max_dim, Matrix& kd_tree, int size, int max_tree_depth, int position){
	
	int left = position*2+1;
	int right = position*2+2;
	
	if(right > size-1 || left > size-1){
		//
		
		kd_tree.elements[position] = sorted_indices[current_dim].elements[0];
		//~ printf("leaf! pos: %d val: %f\n",position, kd_tree.elements[position]);
		
	}else{
		/// split sorted_indices
		int indices_size = sorted_indices[current_dim].width;
		
		// calculate left balanced sizes
		int next_pot = static_cast<int>(log2f(indices_size-1));
		int right_size = pow(2,next_pot-1);
		int left_size = indices_size - right_size;
		int val_next_pot = pow(2,next_pot);
		if( left_size > val_next_pot ){
			right_size += left_size - val_next_pot;
			left_size = val_next_pot;
		}
		
		float split_value = (V.elements[current_dim*V.width+static_cast<int>(sorted_indices[current_dim].elements[left_size-1]) ] + V.elements[current_dim*V.width+static_cast<int>(sorted_indices[current_dim].elements[left_size] ) ] ) /2.0;
		
		kd_tree.elements[position] = split_value;
		
		struct Matrix sorted_indices_left[max_dim];
		struct Matrix sorted_indices_right[max_dim];
		
		// alloc new memory
		for(int i=0; i<max_dim; i++){
			//memory corruption when malloc 
			
			sorted_indices_left[i].width = left_size;
			sorted_indices_left[i].height = 1;
			sorted_indices_left[i].elements = (float*)malloc( (left_size+1) *sizeof(float) );
			
			sorted_indices_right[i].width = right_size;
			sorted_indices_right[i].height = 1;
			sorted_indices_right[i].elements = (float*)malloc( (right_size+1) * sizeof(float) );
			
			if(i==current_dim){
				splitMatrix(sorted_indices[i], sorted_indices_left[i], sorted_indices_right[i]);
			}else{
				splitMatrixWithValue(V, sorted_indices[i], sorted_indices_left[i], sorted_indices_right[i], current_dim, split_value);
			}
			
		}
		
		generateKdTreeRecursive(V, sorted_indices_left, (current_dim+1)%max_dim, max_dim, kd_tree, size, max_tree_depth, left);
		generateKdTreeRecursive(V, sorted_indices_right, (current_dim+1)%max_dim, max_dim, kd_tree, size, max_tree_depth, right);		
	
		
		// alloc new memory
		for(int i=0; i<max_dim; i++){
			free(sorted_indices_left[i].elements);
			free(sorted_indices_right[i].elements);
		}
	}
	
	
}

void generateKdTreeArray(Matrix& V, Matrix* sorted_indices, int max_dim, Matrix& kd_tree, int& size, int& max_tree_depth){
	
	
	max_tree_depth = static_cast<int>(log2f(V.width-1)+2.0) ;
	int max_leaf_size = static_cast<int>(pow(2,max_tree_depth) );
	
	if(V.width == 1){
		max_tree_depth = 1;
	}
	printf("tree depth: %d\n",max_tree_depth);
	
	size = V.width * 2 - 1;
	
	printf("calulated kd-tree size: %d\n",size);
	generateHostMatrix(kd_tree, size, 1);
	
	//start real generate
	generateKdTreeRecursive(V, sorted_indices, 0, max_dim, kd_tree, size, max_tree_depth, 0);
	
	
}

// Get a matrix element
__device__ int GetKdTreePosition(const Matrix& D_kd_tree, float x, float y, float z)
{
	int pos = 0;
	int current_dim = 0;
	
	while(pos*2+1 < D_kd_tree.width)
	{
		
		if(current_dim == 0)
		{
			if(x <= D_kd_tree.elements[pos] )
			{
				pos = pos*2+1;
			} else {
				pos = pos*2+2;
			}
			
			current_dim += 1;
			
		} else if(current_dim == 1) {
			
			if(y <= D_kd_tree.elements[pos] ){
				pos = pos*2+1;
			}else{
				pos = pos*2+2;
			}
			
			current_dim +=1;
		} else {
			if(z <= D_kd_tree.elements[pos] ){
				pos = pos*2+1;
			}else{
				pos = pos*2+2;
			}
			
			current_dim = 0;
		}
		
	}
	
    return pos;
}

__device__ float SearchQueryPoint(const Matrix& D_kd_tree, float x, float y, float z){
	return D_kd_tree.elements[GetKdTreePosition(D_kd_tree, x, y, z)];
}

__device__ void calculateNormalFromSubtree(const Matrix& D_V, const Matrix& D_kd_tree, int pos, int k, float& x,float& y,float& z){
	
	
	int pos_value = (int)(D_kd_tree.elements[pos]+0.5);
	
	
	int subtree_pos = pos;
	
	for(int i=1; i<(k+1) && subtree_pos>0; i*=2){
		subtree_pos = (int)((subtree_pos  - 1) / 2);
	}
	//~ printf("subtree_pos: %d\n",subtree_pos);
	
	int iterator = subtree_pos;
	int max_nodes = 1;
	bool leaf_reached = false;
	int i_nn = 0;
	float * nn_vecs = (float*)malloc(3*k*sizeof(float)+1);

	for(;iterator < D_kd_tree.width; iterator=iterator*2+1, max_nodes*=2)
	{
		for(int i=0; i < max_nodes && iterator + i < D_kd_tree.width; i++)
		{
			int current_pos = iterator+i;
			int leaf_value = (int)(D_kd_tree.elements[current_pos]+0.5);
			
			
			if(leaf_reached && i_nn <= k*3){
				
				
				
				
				if(leaf_value != pos_value){
					//~ printf("index: %d, neighbor_index: %d\n",pos_value,leaf_value);
					//~ printf("tree_index: %d, tree_neighbor_index: %d\n",pos,current_pos);
					nn_vecs[i_nn] = D_V.elements[leaf_value] - D_V.elements[pos_value];
					nn_vecs[i_nn+1] = D_V.elements[D_V.width + leaf_value] - D_V.elements[D_V.width + pos_value];
					nn_vecs[i_nn+2] = D_V.elements[2 * D_V.width + leaf_value] - D_V.elements[2 * D_V.width + pos_value];
					i_nn += 3;
				}
			}else if(current_pos*2+1 >= D_kd_tree.width){
				
				
				//first leaf reached 
				leaf_reached = true;
				if(leaf_value != pos_value && i_nn <= k*3){
					//~ printf("index: %d, neighbor_index: %d\n",pos_value,leaf_value);
					//~ printf("tree_index: %d, tree_neighbor_index: %d\n",pos,current_pos);
					nn_vecs[i_nn] = D_V.elements[leaf_value] - D_V.elements[pos_value];
					nn_vecs[i_nn+1] = D_V.elements[D_V.width + leaf_value] - D_V.elements[D_V.width + pos_value];
					nn_vecs[i_nn+2] = D_V.elements[2 * D_V.width + leaf_value] - D_V.elements[2 * D_V.width + pos_value];
					i_nn += 3;
				}
			}
		}
	}
	
	float * last_vec = (float*)malloc(3 * sizeof(float) );
	last_vec[0] = nn_vecs[0];
	last_vec[1] = nn_vecs[1];
	last_vec[2] = nn_vecs[2];
	float min_dist = FLT_MAX;
	// nearest neighbors in nn!!
	// what now? 
	// PCA?
	// minimize plane error:
	for(int i=3; i<k*3; i+=3){
		// cross product
		
		//~ printf("%f %f %f\n", last_vec[0], last_vec[1], last_vec[2]);
		float n_x = last_vec[1]*nn_vecs[i+2] - last_vec[2]*nn_vecs[i+1];
		float n_y = last_vec[2]*nn_vecs[i+0] - last_vec[0]*nn_vecs[i+2];
		float n_z = last_vec[0]*nn_vecs[i+1] - last_vec[1]*nn_vecs[i+0];
		
		float norm = 1/sqrtf( n_x*n_x + n_y*n_y + n_z*n_z );
		//~ float norm = n_x*n_x + n_y*n_y + n_z*n_z ;
		n_x = n_x * norm;
		n_y = n_y * norm;
		n_z = n_z * norm;
		//~ printf("%f %f %f\n",n_x,n_y,n_z);
		
		float cum_dist = 0.0;
		for(int j=0; j<k*3; j+=3){
			cum_dist += (n_x * nn_vecs[j] + n_y * nn_vecs[j+1] + n_z * nn_vecs[j+2]);
		}
		
		if(cum_dist < min_dist){
			min_dist = cum_dist;
			x = n_x;
			y = n_y;
			z = n_z;
			//~ printf("%f %f %f\n",x,y,z);
		}
		
		last_vec[0] = nn_vecs[i+0];
		last_vec[1] = nn_vecs[i+1];
		last_vec[2] = nn_vecs[i+2];
		
	}
	
	free(last_vec);
	free(nn_vecs);
	
} 

//distance function without transformation
__global__ void KNNKernel(const Matrix D_V, const Matrix D_kd_tree, Matrix D_Result_Normals, int k)
{
	const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(tid < D_V.width){
		
		int pos = GetKdTreePosition(D_kd_tree, D_V.elements[tid], D_V.elements[D_V.width + tid], D_V.elements[D_V.width*2 + tid] );
		
		float val = SearchQueryPoint(D_kd_tree, D_V.elements[tid], D_V.elements[D_V.width + tid], D_V.elements[D_V.width*2 + tid] );
		
		float result_x = 0.0;
		float result_y = 0.0;
		float result_z = 0.0;
		calculateNormalFromSubtree(D_V, D_kd_tree, pos, k, result_x, result_y, result_z);
		
		D_Result_Normals.elements[tid] = result_x;
		D_Result_Normals.elements[D_Result_Normals.width + tid] = result_y;
		D_Result_Normals.elements[D_Result_Normals.width*2 +tid] = result_z;
		
	}
	
}

__global__ void FlipNormalsKernel(const Matrix D_V, Matrix D_Result_Normals, float x, float y, float z){
	const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(tid < D_V.width){
		float x_dir = x - D_V.elements[tid];
		float y_dir = y - D_V.elements[D_V.width + tid];
		float z_dir = z - D_V.elements[2 * D_V.width + tid];
		
		float scalar = ( x_dir * D_Result_Normals.elements[tid] + y_dir * D_Result_Normals.elements[D_Result_Normals.width + tid] + z_dir * D_Result_Normals.elements[2 * D_Result_Normals.width + tid] );
		
		// gegebenfalls < durch > ersetzen
		if(scalar < 0)
		{
			D_Result_Normals.elements[tid] = -D_Result_Normals.elements[tid];
			D_Result_Normals.elements[D_Result_Normals.width + tid] = -D_Result_Normals.elements[D_Result_Normals.width + tid];
			D_Result_Normals.elements[2 * D_Result_Normals.width + tid] = -D_Result_Normals.elements[2 * D_Result_Normals.width + tid];
		}
	}
}


void GPU_NN(Matrix& D_V, Matrix& D_kd_tree, Matrix& D_Result_Normals, Matrix& Result_Normals){
	printf("START\n");
	
	clock_t calcstart, calcend;
	calcstart = clock();
	
	int threadsPerBlock = m_threads_per_block;
	int blocksPerGrid = (D_V.width + threadsPerBlock-1)/threadsPerBlock;

	KNNKernel<<<blocksPerGrid, threadsPerBlock >>>(D_V, D_kd_tree, D_Result_Normals, 50);
	hipDeviceSynchronize();
	FlipNormalsKernel<<<blocksPerGrid, threadsPerBlock >>>(D_V, D_Result_Normals, 1000.0, 1000.0, 1000.0);
	hipDeviceSynchronize();
	
	size_t size = Result_Normals.width * Result_Normals.height * sizeof(float);
	printf("size: %d\n", (int)size);
	
	calcend = clock();
	printf("kNN GPU %f milliseconds\n",(float)(calcend-calcstart)*1000.0 / CLOCKS_PER_SEC);
	
	hipMemcpy(Result_Normals.elements, D_Result_Normals.elements, size, hipMemcpyDeviceToHost);
	
	
	printf("END\n");
}


int main(int argc, char** argv)
{
	getCudaInformation(m_mps, m_cuda_cores_per_mp, m_threads_per_mp, m_threads_per_block, m_size_thread_block, m_size_grid, m_device_global_memory);
	
	const char * filename  = "points.ply";
	
	//HOST STUFF
	int point_dim = 3;
	int num_points = 4;
	
	if(argc > 1){
		num_points = atoi(argv[1]);
	}
	
	int k=50;
	int dim_points = 3;
	
	Matrix V,Result_Normals;
	struct Matrix test;
	struct Matrix indices_sorted[point_dim];
	struct Matrix values_sorted[point_dim];
	
	generateHostMatrix(test, num_points, 1);
	generateHostMatrix( V, num_points, point_dim);
	generateHostMatrix( Result_Normals, num_points, point_dim);
	fillMatrixWithRandomFloats( V);
	//~ fillMatrixWithRandomFloats( Result_Normals);
	
	
	
	for(int i=0; i < point_dim; i++)
	{
		printf("generate indices for dim %d\n",i);
		generateHostMatrix(indices_sorted[i], V.width, 1);
		
		printf("generate values for dim %d\n",i);
		generateHostMatrix(values_sorted[i], V.width,1);
		fillMatrixWithSequence(indices_sorted[i]);
		
		sortByDim( V, i, indices_sorted[i] , values_sorted[i]);
	}
	//~ printMatrix(V);
	
	printf("Start generating kd-tree array based\n");
	//sorted indices + values
	//do some stuff
	Matrix kd_tree;
	int size = 0;
	int max_tree_depth = 0;
	
	generateKdTreeArray(V, indices_sorted, dim_points, kd_tree, size, max_tree_depth);
	
	//~ printMatrix(V);
	printf("End generating kd-tree array based\n");
	
	
	//push values to device
	//push kd_tree to device
	//DEVICE STUFF
	Matrix D_V, D_kd_tree, D_Result_Normals;
	generateDeviceMatrix( D_V, V.width, V.height );
	generateDeviceMatrix( D_kd_tree, kd_tree.width, kd_tree.height);
	generateDeviceMatrix( D_Result_Normals, Result_Normals.width, Result_Normals.height);
	
	//COPY STUFF
	copyToDeviceMatrix( V, D_V);
	copyToDeviceMatrix( kd_tree, D_kd_tree);
	copyToDeviceMatrix( Result_Normals, D_Result_Normals);
	
	
	//Cuda Kernels
	GPU_NN(D_V, D_kd_tree, D_Result_Normals, Result_Normals);
	
	
	
	//~ printMatrix(Result_Normals);
	
	hipFree(D_V.elements);
	hipFree(D_kd_tree.elements);
	hipFree(D_Result_Normals.elements);
	
	
	for(int i=0; i<point_dim;i++)
	{
		printf("free indices dim %d\n",i+1);
		free(indices_sorted[i].elements);
		
		printf("free values dim %d\n",i+1);
		free(values_sorted[i].elements);
	}
	
	free(V.elements);
	free(test.elements);
	free(Result_Normals.elements);
	printf("Free kd_tree array\n");
	free(kd_tree.elements);
}
