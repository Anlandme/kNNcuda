#include "hip/hip_runtime.h"
/*
 * This file is part of cudaNormals.
 *
 * cudaNormals is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * Foobar is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with cudaNormals.  If not, see <http://www.gnu.org/licenses/>.
 */

/**
 * calcNormalsCuda.h
 *
 * @author Alexander Mock
 * @author Matthias Greshake
 */

#include "../include/calcNormalsCuda.h"

/// Define Kernels

__global__ void FlipNormalsKernel(const PointArray D_V, PointArray D_Result_Normals, float x, float y, float z);

__global__ void KNNKernel(const PointArray D_V, const PointArray D_kd_tree, PointArray D_Result_Normals, int k, int method);




__global__ void FlipNormalsKernel(const PointArray D_V, PointArray D_Result_Normals, float x, float y, float z)
{
	const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < D_V.width){
		float x_dir = x - D_V.elements[tid];
		float y_dir = y - D_V.elements[D_V.width + tid];
		float z_dir = z - D_V.elements[2 * D_V.width + tid];
		
		float scalar = ( x_dir * D_Result_Normals.elements[tid] + y_dir * D_Result_Normals.elements[D_Result_Normals.width + tid] + z_dir * D_Result_Normals.elements[2 * D_Result_Normals.width + tid] );
		
		// gegebenfalls < durch > ersetzen
		if(scalar < 0)
		{
			D_Result_Normals.elements[tid] = -D_Result_Normals.elements[tid];
			D_Result_Normals.elements[D_Result_Normals.width + tid] = -D_Result_Normals.elements[D_Result_Normals.width + tid];
			D_Result_Normals.elements[2 * D_Result_Normals.width + tid] = -D_Result_Normals.elements[2 * D_Result_Normals.width + tid];
		}
	}
}

// Get a matrix element
__device__ int GetKdTreePosition(const PointArray& D_kd_tree, float x, float y, float z)
{
	int pos = 0;
	int current_dim = 0;
	
	while(pos*2+1 < D_kd_tree.width)
	{
		
		if(current_dim == 0)
		{
			if(x <= D_kd_tree.elements[pos] )
			{
				pos = pos*2+1;
			} else {
				pos = pos*2+2;
			}
			
			current_dim += 1;
			
		} else if(current_dim == 1) {
			
			if(y <= D_kd_tree.elements[pos] ){
				pos = pos*2+1;
			}else{
				pos = pos*2+2;
			}
			
			current_dim +=1;
		} else {
			if(z <= D_kd_tree.elements[pos] ){
				pos = pos*2+1;
			}else{
				pos = pos*2+2;
			}
			
			current_dim = 0;
		}
		
	}
	
    return pos;
}

__device__ float SearchQueryPoint(const PointArray& D_kd_tree, float x, float y, float z)
{
	return D_kd_tree.elements[GetKdTreePosition(D_kd_tree, x, y, z)];
}


__device__ void calculateNormalRansa2(float* nn_vecs, int k, int max_iterations, float& x, float& y, float& z, bool debug=false){
	float min_dist = FLT_MAX;
	int iterations = 0;
	
	for(int i=3; i<k*3; i+=3){
		//~ printf("%f %f %f\n", last_vec[0], last_vec[1], last_vec[2]);
		
		int j = (i + int(k/3) * 3) % (k * 3);
		
		float n_x = nn_vecs[j+1]*nn_vecs[i+2] - nn_vecs[j+2]*nn_vecs[i+1];
		float n_y = nn_vecs[j+2]*nn_vecs[i+0] - nn_vecs[j+0]*nn_vecs[i+2];
		float n_z = nn_vecs[j+0]*nn_vecs[i+1] - nn_vecs[j+1]*nn_vecs[i+0];
		
		float norm = sqrtf( n_x*n_x + n_y*n_y + n_z*n_z );
		
		
		if( norm != 0.0 ){
			
			float norm_inv = 1.0/norm;
			
			n_x = n_x * norm_inv;
			n_y = n_y * norm_inv;
			n_z = n_z * norm_inv;
			
			float cum_dist = 0.0;
			for(int j=0; j<k*3; j+=3){
				cum_dist += abs(n_x * nn_vecs[j] + n_y * nn_vecs[j+1] + n_z * nn_vecs[j+2]);
			}
			
			if(cum_dist < min_dist) {
				
				iterations = 0;
				min_dist = cum_dist;
				x = n_x;
				y = n_y;
				z = n_z;
				
			} else if(iterations < max_iterations) {
				
				iterations ++;
			
			}else{
				
				return;
				
			}
		}
		
	}
}

__device__ void calculateNormalRansa(float* nn_vecs, int k, int max_iterations, float& x, float& y, float& z, bool debug=false){
	
	float * last_vec = (float*)malloc(3 * sizeof(float) );
	last_vec[0] = nn_vecs[0];
	last_vec[1] = nn_vecs[1];
	last_vec[2] = nn_vecs[2];
	float min_dist = FLT_MAX;
	int iterations = 0;
	// nearest neighbors in nn!!
	// what now? 
	// PCA?
	// minimize plane error:
	
	
	for(int i=3; i<k*3; i+=3){
		// cross product
		
		
		
		//~ printf("%f %f %f\n", last_vec[0], last_vec[1], last_vec[2]);
		float n_x = last_vec[1]*nn_vecs[i+2] - last_vec[2]*nn_vecs[i+1];
		float n_y = last_vec[2]*nn_vecs[i+0] - last_vec[0]*nn_vecs[i+2];
		float n_z = last_vec[0]*nn_vecs[i+1] - last_vec[1]*nn_vecs[i+0];
		
		
		
		
		float norm = sqrtf( n_x*n_x + n_y*n_y + n_z*n_z );
		
		if( norm == 0.0){
				
			last_vec[0] = nn_vecs[i+0];
			last_vec[1] = nn_vecs[i+1];
			last_vec[2] = nn_vecs[i+2];
			continue;
			
		}
		if(debug){
			printf("%d: %f %f %f\n",i,n_x, n_y, n_z);
		}
		
		
		
		
		float norm_inv = 1.0/norm;
		//~ float norm = n_x*n_x + n_y*n_y + n_z*n_z ;
		n_x = n_x * norm_inv;
		n_y = n_y * norm_inv;
		n_z = n_z * norm_inv;
		//~ printf("%f %f %f\n",n_x,n_y,n_z);
		
		float cum_dist = 0.0;
		for(int j=0; j<k*3; j+=3){
			cum_dist += abs(n_x * nn_vecs[j] + n_y * nn_vecs[j+1] + n_z * nn_vecs[j+2]);
		}
		
		if(cum_dist < min_dist){
			iterations = 0;
			min_dist = cum_dist;
			x = n_x;
			y = n_y;
			z = n_z;
			//~ printf("%f %f %f\n",x,y,z);
		}else{
			iterations +=1;
		}
		
		last_vec[0] = nn_vecs[i+0];
		last_vec[1] = nn_vecs[i+1];
		last_vec[2] = nn_vecs[i+2];
		
		if(iterations > max_iterations){
			break;
		}
	}
	
	//instead of minimize plane error:
	// take normal with maximum of inliers (RANSAC like)
	
	free(last_vec);
}

__device__ void calculateNormalPCA(float* nn_vecs, int k, float& n_x, float& n_y, float& n_z){
	
	// ilikebigbits.com/blog/2015/3/2/plane-from-points
	
	
	//x
	float xx = 0.0;
	float xy = 0.0;
	float xz = 0.0;
	
	//y
	float yy = 0.0;
	float yz = 0.0;
	
	//z
	float zz = 0.0;
	
	for(int i=0; i<k; i++)
	{
		float rx = nn_vecs[i*3+0];
		float ry = nn_vecs[i*3+1];
		float rz = nn_vecs[i*3+2];
		
		xx += rx * rx;
		xy += rx * ry;
		xz += rx * rz;
		yy += ry * ry;
		yz += ry * rz;
		zz += rz * rz;
	}
	
	//determinante? 
	float det_x = yy * zz - yz * yz;
	float det_y = xx * zz - xz * xz;
	float det_z = xx * yy - xy * xy;
	
	float dir_x;
	float dir_y;
	float dir_z;
	// det X biggest
	if( det_x >= det_y && det_x >= det_z){
		
		if(det_x <= 0.0){
			//not a plane
		}
		
		dir_x = 1.0;
		dir_y = (xz * yz - xy * zz) / det_x;
		dir_z = (xy * yz - xz * yy) / det_x;
	} //det Y biggest
	else if( det_y >= det_x && det_y >= det_z){
		
		if(det_y <= 0.0){
			// not a plane
		}
		
		dir_x = (yz * xz - xy * zz) / det_y;
		dir_y = 1.0;
		dir_z = (xy * xz - yz * xx) / det_y;
	} // det Z biggest
	else{
		if(det_z <= 0.0){
			// not a plane
		}
		
		dir_x = (yz * xy - xz * yy ) / det_z;
		dir_y = (xz * xy - yz * xx ) / det_z;
		dir_z = 1.0;
	}
	
	float invnorm = 1/sqrtf( dir_x * dir_x + dir_y * dir_y + dir_z * dir_z );
	
	n_x = dir_x * invnorm;
	n_y = dir_y * invnorm;
	n_z = dir_z * invnorm;
	
}

__device__ void switchNeighbor(float* nn_vecs, int k, float v_x, float v_y, float v_z){
	
	if( ( v_x==0.0 || v_x==-0.0 ) &&
			( v_y==0.0 || v_y==-0.0 ) && 
			( v_z==0.0 || v_z==-0.0 ) )
	{
		return;
	}
	
	for(int i=0; i<k*3; i+=3){
		if( ( nn_vecs[i]==0.0 || nn_vecs[i]==-0.0 ) &&
			( nn_vecs[i+1]==0.0 || nn_vecs[i+1]==-0.0 ) && 
			( nn_vecs[i+2]==0.0 || nn_vecs[i+2]==-0.0 ) )
		{
			nn_vecs[i] = v_x;
			nn_vecs[i+1] = v_y;
			nn_vecs[i+2] = v_z;
		} else {
			float dist_old = nn_vecs[i]*nn_vecs[i] + nn_vecs[i+1]*nn_vecs[i+1] + nn_vecs[i+2]*nn_vecs[i+2];
			float dist_new = v_x*v_x + v_y*v_y + v_z*v_z;
			if(dist_new < dist_old){
				nn_vecs[i] = v_x;
				nn_vecs[i+1] = v_y;
				nn_vecs[i+2] = v_z;
			}
		}
	}
}

__device__ void getNearestNeighbors(const PointArray& D_V, const PointArray& D_kd_tree, int k, int subtree_pos, int pos, int pos_value, float* nn_vecs, bool debug=false){
	
	
	int iterator = subtree_pos;
	int max_nodes = 1;
	bool leaf_reached = false;
	int i_nn = 0;
	
	
	for( ;iterator < D_kd_tree.width; iterator=iterator*2+1, max_nodes*=2)
	{
		for( int i=0; i < max_nodes && iterator + i < D_kd_tree.width; i++)
		{
			int current_pos = iterator+i;
			int leaf_value = (int)(D_kd_tree.elements[current_pos]+0.5);
			
			
			if( leaf_reached && i_nn <= k*3 ){
				
				if(leaf_value != pos_value){
					//~ printf("index: %d, neighbor_index: %d\n",pos_value,leaf_value);
					//~ printf("tree_index: %d, tree_neighbor_index: %d\n",pos,current_pos);
					
					float nn_x = D_V.elements[leaf_value * D_V.dim] - D_V.elements[pos_value * D_V.dim];
					float nn_y = D_V.elements[leaf_value * D_V.dim + 1 ] - D_V.elements[pos_value * D_V.dim + 1 ];
					float nn_z = D_V.elements[leaf_value * D_V.dim + 2 ] - D_V.elements[pos_value * D_V.dim + 2 ];
					
					if(nn_x != 0.0 || nn_y != 0.0 || nn_z != 0.0)
					{
						nn_vecs[i_nn] = nn_x;
						nn_vecs[i_nn + 1] = nn_y;
						nn_vecs[i_nn + 2] = nn_z;
					
					
						i_nn += 3;
					}
					
				}
			}else if( current_pos*2+1 >= D_kd_tree.width){
				
				//first leaf reached 
				leaf_reached = true;
				if( leaf_value != pos_value && i_nn <= k*3 ){
					//~ printf("index: %d, neighbor_index: %d\n",pos_value,leaf_value);
					//~ printf("tree_index: %d, tree_neighbor_index: %d\n",pos,current_pos);
					
					
					nn_vecs[i_nn] = D_V.elements[leaf_value * D_V.dim] - D_V.elements[pos_value * D_V.dim ];
					nn_vecs[i_nn+1] = D_V.elements[leaf_value * D_V.dim + 1 ] - D_V.elements[pos_value * D_V.dim + 1 ];
					nn_vecs[i_nn+2] = D_V.elements[leaf_value * D_V.dim + 2] - D_V.elements[pos_value * D_V.dim + 2 ];
					i_nn += 3;
				}
			}
			
			// restlichen values reinschmeissen
			if( i_nn > k * 3){
				if(leaf_value != pos_value){
					//~ printf("index: %d, neighbor_index: %d\n",pos_value,leaf_value);
					//~ printf("tree_index: %d, tree_neighbor_index: %d\n",pos,current_pos);
					//~ float v_x = D_V.elements[leaf_value] - D_V.elements[pos_value];
					//~ float v_y = D_V.elements[D_V.width + leaf_value] - D_V.elements[D_V.width + pos_value];
					//~ float v_z = D_V.elements[2 * D_V.width + leaf_value] - D_V.elements[2 * D_V.width + pos_value];
					
					//~ switchNeighbor(nn_vecs, k, v_x, v_y, v_z);
					
				}
			}
			
		}
	}
	
	if(debug){
		printf("i_nn: %d\n",i_nn);
	}
}

__device__ bool checkLinearNeighborHood(const PointArray& D_V, const PointArray& D_kd_tree, int pos, int k, bool debug=false){
	
	int number_true = 0;
	int * split_positions = (int*)malloc(6*sizeof(int));
	split_positions[0] = (int)((pos  - 1) / 2);
	
	for(int i=1; i<6; i++){
		split_positions[i] = (int)((split_positions[i-1]  - 1) / 2);
		
	}
	
	if(debug){
		int iter = pos;
		for(; iter>0; iter=(int)((iter  - 1) / 2) ){
			printf("%f\n", D_kd_tree.elements[iter ]);
		}
	}
	
	// check x
	for(int i=0;i<3;i++)
	{
		
		if(split_positions[i+3] > 0 )
		{
			if(debug){
				printf("linear check: %f %f\n",D_kd_tree.elements[split_positions[i+3] ], D_kd_tree.elements[split_positions[i] ]);
			}
			if(D_kd_tree.elements[split_positions[i+3] ] != D_kd_tree.elements[split_positions[i] ] )
			{	
				number_true += 1;
			}
		}else{
			number_true += 1;
		}
	}
	
	
	free(split_positions);
	
	if(number_true >= 2){
		return false;
	}else{
		return true;
	}
}

__device__ void calculateNormalFromSubtree(const PointArray& D_V, const PointArray& D_kd_tree, int pos, int k, float& x, float& y, float& z, int method, bool debug=false )
{
	//~ 
	 //~  Step 1: get upper node
	 //~  Step 2: get child nodes != query node
	 //~  Step 3: calculate normals
	 //~ 
	
	bool linear = checkLinearNeighborHood(D_V, D_kd_tree, pos, k, debug);
	
	if(debug){
		if(linear){
			printf("linear! \n");
		}else{
			printf("good! \n");
		}
	}
	
	int pos_value = (int)(D_kd_tree.elements[pos]+0.5);
	
	int subtree_pos = pos;
	int i;
	for(i=1; i<(k+1) && subtree_pos>0; i*=2) {
		subtree_pos = (int)((subtree_pos  - 1) / 2);
	}
	//~ printf("subtree_pos: %d\n",subtree_pos);
	
	
	float * nn_vecs = (float*)malloc(3*k*sizeof(float));
	
	
	getNearestNeighbors(D_V, D_kd_tree, k, subtree_pos, pos, pos_value, nn_vecs, debug); 

	if(method == 0){
		//PCA
		calculateNormalPCA(nn_vecs, k, x, y, z);
	}else if(method == 1) {
		//RANSAC
		calculateNormalRansa2(nn_vecs, k, 8, x, y, z, debug);
	}
	
	free(nn_vecs);
	
} 

//distance function without transformation
__global__ void KNNKernel(const PointArray D_V, const PointArray D_kd_tree, PointArray D_Result_Normals, int k, int method)
{
	const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(tid < D_V.width){
		
		int pos = GetKdTreePosition(D_kd_tree, D_V.elements[tid * D_V.dim], D_V.elements[tid * D_V.dim + 1], D_V.elements[tid * D_V.dim +2] );
		
		float result_x = D_Result_Normals.elements[tid * D_Result_Normals.dim ];
		float result_y = D_Result_Normals.elements[tid * D_Result_Normals.dim + 1 ];
		float result_z = D_Result_Normals.elements[tid * D_Result_Normals.dim + 2 ];
		// no normal at 68088996
		bool debug=false;
		//~ if(tid == 10000){
			//~ debug = true;
		//~ }
		
		calculateNormalFromSubtree(D_V, D_kd_tree, pos, k, result_x, result_y, result_z, method, debug);
		
		D_Result_Normals.elements[tid * D_Result_Normals.dim ] = result_x;
		D_Result_Normals.elements[tid * D_Result_Normals.dim + 1 ] = result_y;
		D_Result_Normals.elements[tid * D_Result_Normals.dim + 2 ] = result_z;
		
	}
	
}

void CalcNormalsCuda::init(){
	// set default k
	this->m_k = 50;
	
	// set default flippoint
	this->m_vx = 1000000.0;
	this->m_vy = 1000000.0;
	this->m_vz = 1000000.0;
	
	this->m_calc_method = 0;
}

CalcNormalsCuda::CalcNormalsCuda(PointArray& points)
{
	printf("Constructing kd-tree...\n");
	this->init();
	
	CalcNormalsCuda::getCudaInformation();
	
	this->V.dim = points.dim;
	
	this->V.width = points.width;
	
	mallocPointArray(V);
	
	for(int i = 0; i<points.width*points.dim; i++)
	{
		
		this->V.elements[i] = points.elements[i];
		
	}
	
	this->initKdTree();
	
}

void CalcNormalsCuda::getCudaInformation()
{
	
	m_mps = 0;
	m_threads_per_mp = 0;
	m_threads_per_block = 0;
	m_size_thread_block = new int(3);
	m_size_grid = new int(3);
	m_device_global_memory = 0;
	
	
	hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    
    
    m_mps = deviceProp.multiProcessorCount;
    m_threads_per_mp = deviceProp.maxThreadsPerMultiProcessor;
    m_threads_per_block = deviceProp.maxThreadsPerBlock;
    m_size_thread_block[0] = deviceProp.maxThreadsDim[0];
    m_size_thread_block[1] = deviceProp.maxThreadsDim[1];
    m_size_thread_block[2] = deviceProp.maxThreadsDim[2];
    m_size_grid[0] = deviceProp.maxGridSize[0];
    m_size_grid[1] = deviceProp.maxGridSize[1];
    m_size_grid[2] = deviceProp.maxGridSize[2];
    m_device_global_memory = (unsigned long long) deviceProp.totalGlobalMem;
    
}

void CalcNormalsCuda::getNormals(PointArray& output_normals)
{
	
	output_normals.dim = this->Result_Normals.dim;
	output_normals.width = this->Result_Normals.width;
	output_normals.elements = (float*)malloc( this->Result_Normals.dim * this->Result_Normals.width * sizeof(float) ) ;
	
	for(int i = 0; i< this->Result_Normals.dim * this->Result_Normals.width; i++)
	{	
		output_normals.elements[i] = this->Result_Normals.elements[i];
	}
	
}

void CalcNormalsCuda::mallocPointArray(PointArray& m)
{
	m.elements = (float*)malloc(m.width * m.dim * sizeof(float));
}

void CalcNormalsCuda::generateHostPointArray(PointArray& m, int width, int dim)
{
	
	m.dim = dim;
	m.width = width;
	m.elements = (float*)malloc(m.width * m.dim * sizeof(float) );
	
}

void CalcNormalsCuda::generateDevicePointArray(PointArray& D_m, int width, int dim){
	
    D_m.width = width;
    D_m.dim = dim;
    size_t size = D_m.width * D_m.dim * sizeof(float);
    hipMalloc(&D_m.elements, size);
    
}

void CalcNormalsCuda::copyToDevicePointArray(PointArray& m, PointArray& D_m){
	
	size_t size = m.width * m.dim * sizeof(float);
    hipMemcpy(D_m.elements, m.elements, size, hipMemcpyHostToDevice);

}

void CalcNormalsCuda::copyToHostPointArray(PointArray& D_m, PointArray& m){
	
	size_t size = m.width * m.dim * sizeof(float);
	hipMemcpy(m.elements, D_m.elements, size, hipMemcpyDeviceToHost);
	
}

void CalcNormalsCuda::fillPointArrayWithSequence(PointArray& m)
{
	for(int i=0;i<m.width*m.dim;i++)
	{
		m.elements[i] = i;
	}
}  

void CalcNormalsCuda::copyDimensionToPointArray(PointArray& in, int dim, PointArray& out)
{
	for(int i = 0; i<out.width; i++)
	{	
		out.elements[i] = in.elements[i * in.dim + dim];
	}
}

void CalcNormalsCuda::copyVectorInterval(PointArray& in, int start, int end, PointArray& out){
	for(int i=0; i < (end-start); i++){
		out.elements[i] = in.elements[i + start];
	}
}

void CalcNormalsCuda::mergeHostWithIndices(float* a, float* b, int i1, int j1, int i2, int j2, int limit)
{
	int limit_end = limit;
	
	float* temp = (float*) malloc((j2-i1+1) * sizeof(float));  //array used for merging
    int* temp_indices = (int*) malloc((j2-i1+1) * sizeof(int));  //array used for merging
    
    
    int i,j,k;
    i=i1;    //beginning of the first list
    j=i2;    //beginning of the second list
    k=0;
    
    int counter = 0;
    while(i<=j1 && j<=j2 && limit!=0)    //while elements in both lists
    {
		counter ++;
		limit--;
        if(a[i]<a[j]){
			temp_indices[k] = b[i]; 
            temp[k++]=a[i++];
            
        }else{
			temp_indices[k] = b[j];
            temp[k++]=a[j++];
		}
    }
    
    while(i<=j1 && limit!=0) {   //copy remaining elements of the first list
		temp_indices[k] = b[i]; 
        temp[k++]=a[i++];
	}
        
    while(j<=j2 && limit!=0) {   //copy remaining elements of the second list
        temp_indices[k] = b[j]; 
        temp[k++]=a[j++];
	}
        
    //Transfer elements from temp[] back to a[]
    for(i=i1,j=0;i<=j2 && limit_end!=0 ;i++,j++,limit_end--)
	{
		b[i] = temp_indices[j];
		if(b[i] < 0){
			printf("THERE IS SOMETHING WRONG\n");
		}
        a[i] = temp[j];
    }   
    free(temp_indices);
    free(temp);
}


void CalcNormalsCuda::naturalMergeSort(PointArray& in, int dim, PointArray& indices, PointArray& m, int limit){
	
	copyDimensionToPointArray(in, dim, m);
	//~ printf("copy row of Mat for dim %d\n", dim);
	//~ printMatrix(m,true);
	
	int m_elements = m.width * m.dim;
	
	int slide_buffer_size = int(m_elements-0.5);
	int* slide_buffer = (int*) malloc(slide_buffer_size * sizeof(int));
	

	//create RUNS
	int num_slides = 1;
	slide_buffer[0] = 0;
	for(int i=1; i < slide_buffer_size+1; i++) 
	{
		if(m.elements[i] < m.elements[i-1])
		{
			slide_buffer[num_slides] = i;
			num_slides++;
		}
		
	}
	slide_buffer[num_slides] = m_elements;
	slide_buffer_size = num_slides+1;
	
	
	//sort 
	int count = 0;
	int current_limit = -1;
	while(num_slides > 1)
	{
		if(num_slides > 2){
			current_limit = limit;
		}
		
		int i;
		
		for(i=2;i<int(num_slides+1);i+=2)
		{
				
			mergeHostWithIndices(m.elements, indices.elements , slide_buffer[i-2], slide_buffer[i-1]-1, slide_buffer[i-1], slide_buffer[i]-1, current_limit);
			
			
			slide_buffer[i/2]= slide_buffer[i];
		}
		
		if(num_slides%2 == 1){
			slide_buffer[(num_slides+1)/2] = slide_buffer[num_slides];
		}
		
		count ++;
		num_slides = int(num_slides/2.0+0.5);
		
	}
	
	free(slide_buffer);
}

void CalcNormalsCuda::sortByDim(PointArray& V, int dim, PointArray& indices, PointArray& values)
{
	naturalMergeSort(V, dim, indices, values);
}

void CalcNormalsCuda::splitPointArray(PointArray& I, PointArray& I_L, PointArray& I_R)
{
	
	int i=0;
	for(; i < I_L.width * I_L.dim; i++){
		I_L.elements[i] = I.elements[i];
	}
	int j=0;
	for(; i<I.width*I.dim && j<I_R.width*I_R.dim; i++, j++){
		I_R.elements[j] = I.elements[i];
	}
	
}

void CalcNormalsCuda::splitPointArrayWithValue(PointArray& V, PointArray& I, PointArray& I_L, PointArray& I_R, int current_dim, float value)
{
	int i_l = 0;
	int i_r = 0;
	
	for(int i=0; i<I.width; i++)
	{
		float current_value = V.elements[static_cast<int>(I.elements[i] + 0.5) * V.dim + current_dim ];
		//~ printf("curr val: %f\n", current_value);
		if(current_value <= value && I_L.width > i_l ){
			//~ printf("add to left: %f with value %f\n", I.elements[i], current_value);
			I_L.elements[i_l++] = I.elements[i];
		}else if(current_value >= value && I_R.width > i_r){
			//~ printf("add to right: %f with value %f\n", I.elements[i], current_value);
			I_R.elements[i_r++] = I.elements[i];
		}else {
			if(i_r<I_R.width){
				I_R.elements[i_r++] = I.elements[i];
			}else if(i_l<I_L.width){
				I_L.elements[i_l++] = I.elements[i];
			}
		}
	}
	
	if(i_l != I_L.width){
		printf("WARNING left %d != %d\n",i_l,I_L.width);
	}
	
	if(i_r != I_R.width){
		printf("WARNING right %d != %d\n",i_r,I_R.width);
	}
		
}


void CalcNormalsCuda::generateKdTreeRecursive(PointArray& V, PointArray* sorted_indices, int current_dim, int max_dim, PointArray& kd_tree, int size, int max_tree_depth, int position){
	
	int left = position*2+1;
	int right = position*2+2;
	
	if(right > size-1 || left > size-1){
		//
		
		kd_tree.elements[position] = sorted_indices[current_dim].elements[0];
		//~ printf("leaf! pos: %d val: %f\n",position, kd_tree.elements[position]);
		
	}else{
		/// split sorted_indices
		int indices_size = sorted_indices[current_dim].width;
		
		int v = pow(2,static_cast<int>(log2f(indices_size-1) ) );
		int left_size = indices_size - v/2;
		if( left_size > v ) {
			left_size = v;
		}
		int right_size = indices_size - left_size;
		
		float split_value = (V.elements[current_dim+static_cast<int>(sorted_indices[current_dim].elements[left_size-1])*V.dim ] + V.elements[current_dim+static_cast<int>(sorted_indices[current_dim].elements[left_size] ) * V.dim] ) /2.0;
		
		kd_tree.elements[position] = split_value;
		
		struct PointArray sorted_indices_left[max_dim];
		struct PointArray sorted_indices_right[max_dim];
		
		// alloc new memory
		for(int i=0; i<max_dim; i++){
			//memory corruption when malloc 
			
			sorted_indices_left[i].width = left_size;
			sorted_indices_left[i].dim = 1;
			sorted_indices_left[i].elements = (float*)malloc( (left_size+1) *sizeof(float) );
			
			sorted_indices_right[i].width = right_size;
			sorted_indices_right[i].dim = 1;
			sorted_indices_right[i].elements = (float*)malloc( (right_size+1) * sizeof(float) );
			
			if(i==current_dim){
				splitPointArray( sorted_indices[i], sorted_indices_left[i], sorted_indices_right[i]);
			}else{
				splitPointArrayWithValue(V, sorted_indices[i], sorted_indices_left[i], sorted_indices_right[i], current_dim, split_value);
			}
			
		}
		
		generateKdTreeRecursive(V, sorted_indices_left, (current_dim+1)%max_dim, max_dim, kd_tree, size, max_tree_depth, left);
		generateKdTreeRecursive(V, sorted_indices_right, (current_dim+1)%max_dim, max_dim, kd_tree, size, max_tree_depth, right);		
	
		
		// alloc new memory
		for(int i=0; i<max_dim; i++){
			free(sorted_indices_left[i].elements);
			free(sorted_indices_right[i].elements);
		}
	}
	
	
}

void CalcNormalsCuda::generateKdTreeArray(PointArray& V, PointArray* sorted_indices, int max_dim, PointArray& kd_tree)
{
	int size;
	int max_tree_depth;
	
	max_tree_depth = static_cast<int>( log2f(V.width - 1 ) + 2.0 ) ;
	int max_leaf_size = static_cast<int>( pow(2, max_tree_depth) ) ;
	
	if (V.width == 1)
	{
		max_tree_depth = 1;
	}
	
	//~ printf("tree depth: %d\n",max_tree_depth);
	
	size = V.width * 2 - 1;
	
	//~ printf("calulated kd-tree size: %d\n",size);
	generateHostPointArray(kd_tree, size, 1);
	
	//start real generate
	generateKdTreeRecursive(V, sorted_indices, 0, max_dim, kd_tree, size, max_tree_depth, 0);
	
	
}

void CalcNormalsCuda::GPU_NN(PointArray& D_V, PointArray& D_kd_tree, PointArray& D_Result_Normals, PointArray& Result_Normals)
{
	
	//~ clock_t calcstart, calcend;
	//~ calcstart = clock();
	
	int threadsPerBlock = this->m_threads_per_block;
	int blocksPerGrid = (D_V.width + threadsPerBlock-1)/threadsPerBlock;

	KNNKernel<<<blocksPerGrid, threadsPerBlock >>>(D_V, D_kd_tree, D_Result_Normals, this->m_k, this->m_calc_method);
	hipDeviceSynchronize();
	FlipNormalsKernel<<<blocksPerGrid, threadsPerBlock >>>(D_V, D_Result_Normals, this->m_vx, this->m_vy, this->m_vz);
	hipDeviceSynchronize();
	
	size_t size = Result_Normals.width * Result_Normals.dim * sizeof(float);
	//~ printf("size: %d\n", (int)size);
	
	//~ calcend = clock();
	//~ printf("kNN GPU %f milliseconds\n",(float)(calcend-calcstart)*1000.0 / CLOCKS_PER_SEC);
	
	hipMemcpy(Result_Normals.elements, D_Result_Normals.elements, size, hipMemcpyDeviceToHost );
	
	
	//~ printf("END\n");
}

void CalcNormalsCuda::initKdTree(){
	
	//~ struct Matrix test;
	struct PointArray indices_sorted[this->V.dim];
	struct PointArray values_sorted[this->V.dim];
	
	for(int i=0; i < this->V.dim; i++)
	{
		//~ printf("generate indices for dim %d\n",i);
		generateHostPointArray(indices_sorted[i], V.width, 1);
		
		//~ printf("generate values for dim %d\n",i);
		generateHostPointArray(values_sorted[i], V.width, 1);
		
		fillPointArrayWithSequence(indices_sorted[i]);
		
		sortByDim( this->V, i, indices_sorted[i] , values_sorted[i]);
	}
	
	
	
	
	//~ printf("Start generating kd-tree array based\n");
	
	
	generateKdTreeArray(V, indices_sorted, this->V.dim, this->kd_tree);
	
	//~ printMatrix(V);
	//~ printf("End generating kd-tree array based\n");
	
	//push values to device
	//push kd_tree to device
	//DEVICE STUFF
	
	for(int i=0; i<V.dim;i++)
	{
		free(indices_sorted[i].elements);
		free(values_sorted[i].elements);
	}
}

void CalcNormalsCuda::setK(int k)
{
	this->m_k = k;
}

void CalcNormalsCuda::setFlippoint(float v_x, float v_y, float v_z)
{
	this->m_vx = v_x;
	this->m_vy = v_y;
	this->m_vz = v_z;
}

void CalcNormalsCuda::setMethod(std::string method)
{
	if( strcmp( method.c_str(), "PCA") == 0 ){
		this->m_calc_method = 0;
	} else if( strcmp( method.c_str(), "RANSAC") == 0){
		this->m_calc_method = 1;
	} else {
		printf("WARNING: Normal Calculation Method is not implemented\n");
	}
}

void CalcNormalsCuda::printSettings()
{
	printf("	Nearest Neighbors = %d\n",this->m_k);
	
	printf("	Flip point = (%f, %f, %f)\n", this->m_vx, this->m_vy, this->m_vz);
	
	switch(this->m_calc_method){
		case 0:
			printf("	Method = 'PCA'\n");
			break;
		case 1:
			printf("	Method = 'RANSAC'\n");
			break;
	}
	
	printf("\n");
}

void CalcNormalsCuda::start()
{
	printf("\nStart Normal Calculation\n\n");
	
	printSettings();
	
	
	generateHostPointArray( this->Result_Normals, V.width, V.dim);
	
	PointArray D_V, D_kd_tree, D_Result_Normals;
	generateDevicePointArray( D_V, this->V.width, this->V.dim );
	generateDevicePointArray( D_kd_tree, this->kd_tree.width, this->kd_tree.dim);
	generateDevicePointArray( D_Result_Normals, this->Result_Normals.width, this->Result_Normals.dim);
	
	//COPY STUFF
	copyToDevicePointArray( V, D_V);
	copyToDevicePointArray( this->kd_tree, D_kd_tree);
	//~ copyToDevicePointArray( this->Result_Normals, D_Result_Normals);
	
	
	//Cuda Kernels
	GPU_NN(D_V, D_kd_tree, D_Result_Normals, this->Result_Normals);
	
	hipFree(D_V.elements);
	hipFree(D_kd_tree.elements);
	hipFree(D_Result_Normals.elements);
	
	
}

CalcNormalsCuda::~CalcNormalsCuda(){
	
	if(this->Result_Normals.width > 0){
		free(Result_Normals.elements);
	}
	
	if(this->kd_tree.width > 0){
		free(this->kd_tree.elements);
	}
}


