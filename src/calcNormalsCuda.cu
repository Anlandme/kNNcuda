#include "hip/hip_runtime.h"
/*
 * This file is part of cudaNormals.
 *
 * cudaNormals is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * Foobar is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with cudaNormals.  If not, see <http://www.gnu.org/licenses/>.
 */

/**
 * calcNormalsCuda.h
 *
 * @author Alexander Mock
 * @author Matthias Greshake
 */

#include "calcNormalsCuda.h"

/// Define Kernels

__global__ void FlipNormalsKernel(const PointArray D_V, PointArray D_Result_Normals, float x, float y, float z);

__global__ void KNNKernel(const PointArray D_V, const PointArray D_kd_tree, PointArray D_Result_Normals, int k, int method);




__global__ void FlipNormalsKernel(const PointArray D_V, PointArray D_Result_Normals, float x, float y, float z)
{
	const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < D_V.width){
		float x_dir = x - D_V.elements[tid];
		float y_dir = y - D_V.elements[D_V.width + tid];
		float z_dir = z - D_V.elements[2 * D_V.width + tid];
		
		float scalar = ( x_dir * D_Result_Normals.elements[tid] + y_dir * D_Result_Normals.elements[D_Result_Normals.width + tid] + z_dir * D_Result_Normals.elements[2 * D_Result_Normals.width + tid] );
		
		// gegebenfalls < durch > ersetzen
		if(scalar < 0)
		{
			D_Result_Normals.elements[tid] = -D_Result_Normals.elements[tid];
			D_Result_Normals.elements[D_Result_Normals.width + tid] = -D_Result_Normals.elements[D_Result_Normals.width + tid];
			D_Result_Normals.elements[2 * D_Result_Normals.width + tid] = -D_Result_Normals.elements[2 * D_Result_Normals.width + tid];
		}
	}
}

// Get a matrix element
__device__ int GetKdTreePosition(const PointArray& D_kd_tree, float x, float y, float z)
{
	int pos = 0;
	int current_dim = 0;
	
	while(pos*2+1 < D_kd_tree.width)
	{
		
		if(current_dim == 0)
		{
			if(x <= D_kd_tree.elements[pos] )
			{
				pos = pos*2+1;
			} else {
				pos = pos*2+2;
			}
			
			current_dim += 1;
			
		} else if(current_dim == 1) {
			
			if(y <= D_kd_tree.elements[pos] ){
				pos = pos*2+1;
			}else{
				pos = pos*2+2;
			}
			
			current_dim +=1;
		} else {
			if(z <= D_kd_tree.elements[pos] ){
				pos = pos*2+1;
			}else{
				pos = pos*2+2;
			}
			
			current_dim = 0;
		}
		
	}
	
    return pos;
}

__device__ float SearchQueryPoint(const PointArray& D_kd_tree, float x, float y, float z)
{
	return D_kd_tree.elements[GetKdTreePosition(D_kd_tree, x, y, z)];
}


__device__ void calculateNormalRansa2(float* nn_vecs, int k, int max_iterations, float& x, float& y, float& z){
	float min_dist = FLT_MAX;
	int iterations = 0;
	
	for(int i=3; i<k*3; i+=3){
		//~ printf("%f %f %f\n", last_vec[0], last_vec[1], last_vec[2]);
		
		int j = (i + int(k/3) * 3) % (k * 3);
		
		float n_x = nn_vecs[j+1]*nn_vecs[i+2] - nn_vecs[j+2]*nn_vecs[i+1];
		float n_y = nn_vecs[j+2]*nn_vecs[i+0] - nn_vecs[j+0]*nn_vecs[i+2];
		float n_z = nn_vecs[j+0]*nn_vecs[i+1] - nn_vecs[j+1]*nn_vecs[i+0];
		
		float norm = sqrtf( n_x*n_x + n_y*n_y + n_z*n_z );
		
		
		if( norm != 0.0 ){
			
			float norm_inv = 1.0/norm;
			
			n_x = n_x * norm_inv;
			n_y = n_y * norm_inv;
			n_z = n_z * norm_inv;
			
			float cum_dist = 0.0;
			for(int j=0; j<k*3; j+=3){
				cum_dist += abs(n_x * nn_vecs[j] + n_y * nn_vecs[j+1] + n_z * nn_vecs[j+2]);
			}
			
			if(cum_dist < min_dist) {
				
				iterations = 0;
				min_dist = cum_dist;
				x = n_x;
				y = n_y;
				z = n_z;
				
			} else if(iterations < max_iterations) {
				
				iterations ++;
			
			}else{
				
				return;
				
			}
		}
		
	}
}

__device__ void calculateNormalRansa(float* nn_vecs, int k, int max_iterations, float& x, float& y, float& z){
	
	float * last_vec = (float*)malloc(3 * sizeof(float) );
	last_vec[0] = nn_vecs[0];
	last_vec[1] = nn_vecs[1];
	last_vec[2] = nn_vecs[2];
	float min_dist = FLT_MAX;
	int iterations = 0;
	// nearest neighbors in nn!!
	// what now? 
	// PCA?
	// minimize plane error:
	
	
	for(int i=3; i<k*3; i+=3){
		// cross product
		
		float n_x = last_vec[1]*nn_vecs[i+2] - last_vec[2]*nn_vecs[i+1];
		float n_y = last_vec[2]*nn_vecs[i+0] - last_vec[0]*nn_vecs[i+2];
		float n_z = last_vec[0]*nn_vecs[i+1] - last_vec[1]*nn_vecs[i+0];

		float norm = sqrtf( n_x*n_x + n_y*n_y + n_z*n_z );
		
		if( norm == 0.0){
				
			last_vec[0] = nn_vecs[i+0];
			last_vec[1] = nn_vecs[i+1];
			last_vec[2] = nn_vecs[i+2];
			continue;
			
        }
		
		float norm_inv = 1.0/norm;
		//~ float norm = n_x*n_x + n_y*n_y + n_z*n_z ;
		n_x = n_x * norm_inv;
		n_y = n_y * norm_inv;
		n_z = n_z * norm_inv;
		//~ printf("%f %f %f\n",n_x,n_y,n_z);
		
		float cum_dist = 0.0;
		for(int j=0; j<k*3; j+=3){
			cum_dist += abs(n_x * nn_vecs[j] + n_y * nn_vecs[j+1] + n_z * nn_vecs[j+2]);
		}
		
		if(cum_dist < min_dist){
			iterations = 0;
			min_dist = cum_dist;
			x = n_x;
			y = n_y;
			z = n_z;
			//~ printf("%f %f %f\n",x,y,z);
		}else{
			iterations +=1;
		}
		
		last_vec[0] = nn_vecs[i+0];
		last_vec[1] = nn_vecs[i+1];
		last_vec[2] = nn_vecs[i+2];
		
        if( iterations > max_iterations){
			break;
		}
	}
	
	//instead of minimize plane error:
	// take normal with maximum of inliers (RANSAC like)
	
	free(last_vec);
}

__device__ void calculateNormalPCA(float* nn_vecs, int k, float& n_x, float& n_y, float& n_z){
	
	// ilikebigbits.com/blog/2015/3/2/plane-from-points
	
	
	//x
	float xx = 0.0;
	float xy = 0.0;
	float xz = 0.0;
	
	//y
	float yy = 0.0;
	float yz = 0.0;
	
	//z
	float zz = 0.0;
	
	for(int i=0; i<k; i++)
	{
		float rx = nn_vecs[i*3+0];
		float ry = nn_vecs[i*3+1];
		float rz = nn_vecs[i*3+2];
		
		xx += rx * rx;
		xy += rx * ry;
		xz += rx * rz;
		yy += ry * ry;
		yz += ry * rz;
		zz += rz * rz;
	}
	
	//determinante? 
	float det_x = yy * zz - yz * yz;
	float det_y = xx * zz - xz * xz;
	float det_z = xx * yy - xy * xy;
	
	float dir_x;
	float dir_y;
	float dir_z;
	// det X biggest
	if( det_x >= det_y && det_x >= det_z){
		
		if(det_x <= 0.0){
			//not a plane
		}
		
		dir_x = 1.0;
		dir_y = (xz * yz - xy * zz) / det_x;
		dir_z = (xy * yz - xz * yy) / det_x;
	} //det Y biggest
	else if( det_y >= det_x && det_y >= det_z){
		
		if(det_y <= 0.0){
			// not a plane
		}
		
		dir_x = (yz * xz - xy * zz) / det_y;
		dir_y = 1.0;
		dir_z = (xy * xz - yz * xx) / det_y;
	} // det Z biggest
	else{
		if(det_z <= 0.0){
			// not a plane
		}
		
		dir_x = (yz * xy - xz * yy ) / det_z;
		dir_y = (xz * xy - yz * xx ) / det_z;
		dir_z = 1.0;
	}
	
	float invnorm = 1/sqrtf( dir_x * dir_x + dir_y * dir_y + dir_z * dir_z );
	
	n_x = dir_x * invnorm;
	n_y = dir_y * invnorm;
	n_z = dir_z * invnorm;
	
}

__device__ void switchNeighbor(float* nn_vecs, int k, float v_x, float v_y, float v_z){
	
	if( ( v_x==0.0 || v_x==-0.0 ) &&
			( v_y==0.0 || v_y==-0.0 ) && 
			( v_z==0.0 || v_z==-0.0 ) )
	{
		return;
	}
	
	for(int i=0; i<k*3; i+=3){
		if( ( nn_vecs[i]==0.0 || nn_vecs[i]==-0.0 ) &&
			( nn_vecs[i+1]==0.0 || nn_vecs[i+1]==-0.0 ) && 
			( nn_vecs[i+2]==0.0 || nn_vecs[i+2]==-0.0 ) )
		{
			nn_vecs[i] = v_x;
			nn_vecs[i+1] = v_y;
			nn_vecs[i+2] = v_z;
		} else {
			float dist_old = nn_vecs[i]*nn_vecs[i] + nn_vecs[i+1]*nn_vecs[i+1] + nn_vecs[i+2]*nn_vecs[i+2];
			float dist_new = v_x*v_x + v_y*v_y + v_z*v_z;
            if(dist_new < dist_old) {
				nn_vecs[i] = v_x;
				nn_vecs[i+1] = v_y;
				nn_vecs[i+2] = v_z;
			}
		}
	}
}

__device__ void getNearestNeighbors(const PointArray& D_V,
                                    const PointArray& D_kd_tree,
                                    int k,
                                    int subtree_pos,
                                    int pos,
                                    int pos_value,
                                    float* nn_vecs )
{
	
	
	int iterator = subtree_pos;
	int max_nodes = 1;
	bool leaf_reached = false;
	int i_nn = 0;

    int query_index = pos_value * D_V.dim;

    float query_x = D_V.elements[ query_index ];
    float query_y = D_V.elements[ query_index + 1 ];
    float query_z = D_V.elements[ query_index + 2 ];

    // like width search
    // go kd-tree up until max_nodes(leaf_nodes of subtree) bigger than needed nodes k
    // iterator = iterator * 2 + 1 -> go to
    for( ; iterator < D_kd_tree.width; iterator = iterator * 2 + 1, max_nodes *= 2)
	{
        // collect nodes from current height
		for( int i=0; i < max_nodes && iterator + i < D_kd_tree.width; i++)
		{
            int current_pos = iterator + i;
            int leaf_value  = (int)(D_kd_tree.elements[ current_pos ] + 0.5 );
			
            if( leaf_reached && i_nn <= k*3 )
            {
                if( leaf_value != pos_value && leaf_value < D_V.width )
                {
                    int curr_nn_index = leaf_value * D_V.dim;

                    float nn_x = D_V.elements[ curr_nn_index ] - query_x;
                    float nn_y = D_V.elements[ curr_nn_index + 1 ] - query_y;
                    float nn_z = D_V.elements[ curr_nn_index + 2 ] - query_z;
					
					if(nn_x != 0.0 || nn_y != 0.0 || nn_z != 0.0)
					{
                        nn_vecs[ i_nn ]     = nn_x;
                        nn_vecs[ i_nn + 1 ] = nn_y;
                        nn_vecs[ i_nn + 2 ] = nn_z;
					
						i_nn += 3;
                    }
				}
            } else if( current_pos * 2 + 1 >= D_kd_tree.width ) {
				
                int curr_nn_index = leaf_value * D_V.dim;
				//first leaf reached 
				leaf_reached = true;
                if( leaf_value != pos_value && i_nn <= k*3 )
                {
                    nn_vecs[i_nn]   = D_V.elements[ curr_nn_index ] - query_x;
                    nn_vecs[i_nn+1] = D_V.elements[ curr_nn_index + 1 ] - query_y;
                    nn_vecs[i_nn+2] = D_V.elements[ curr_nn_index + 2] - query_z;
					i_nn += 3;
				}
            }
		}
    }
}

__device__ bool checkLinearNeighborHood(const PointArray& D_V,
                                        const PointArray& D_kd_tree,
                                        int pos,
                                        int k )
{
	int number_true = 0;
	int * split_positions = (int*)malloc(6*sizeof(int));
	split_positions[0] = (int)((pos  - 1) / 2);
	
	for(int i=1; i<6; i++){
		split_positions[i] = (int)((split_positions[i-1]  - 1) / 2);
	}
	
	// check x
    for(int i=0; i<3; i++)
	{
		
		if(split_positions[i+3] > 0 )
		{

			if(D_kd_tree.elements[split_positions[i+3] ] != D_kd_tree.elements[split_positions[i] ] )
			{	
				number_true += 1;
			}
		}else{
			number_true += 1;
		}
	}
	
	
	free(split_positions);
	
	if(number_true >= 2){
		return false;
	}else{
		return true;
	}
}

__device__ void calculateNormalFromSubtree(const PointArray& D_V,
                                           const PointArray& D_kd_tree,
                                           int pos,
                                           int k,
                                           float& x, float& y, float& z,
                                           int method )
{
	//~ 
	 //~  Step 1: get upper node
	 //~  Step 2: get child nodes != query node
	 //~  Step 3: calculate normals
	 //~ 
	
    //bool linear = checkLinearNeighborHood(D_V, D_kd_tree, pos, k);
	
	int pos_value = (int)(D_kd_tree.elements[pos]+0.5);
	
	int subtree_pos = pos;
	int i;
	for(i=1; i<(k+1) && subtree_pos>0; i*=2) {
		subtree_pos = (int)((subtree_pos  - 1) / 2);
	}
	//~ printf("subtree_pos: %d\n",subtree_pos);
	
    // k+1 FIX
    float * nn_vecs = (float*)malloc(3*(k+1)*sizeof(float));
	
	
    getNearestNeighbors(D_V, D_kd_tree, k, subtree_pos, pos, pos_value, nn_vecs);

	if(method == 0){
		//PCA
		calculateNormalPCA(nn_vecs, k, x, y, z);
	}else if(method == 1) {
		//RANSAC
        calculateNormalRansa2(nn_vecs, k, 8, x, y, z);
	}
	
	free(nn_vecs);
	
} 

//distance function without transformation
__global__ void KNNKernel(const PointArray D_V, const PointArray D_kd_tree, PointArray D_Result_Normals, int k, int method)
{
	const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

	const unsigned int query_x = tid * D_Result_Normals.dim;
	if(query_x + 2 < D_Result_Normals.width * D_Result_Normals.dim){
		
		int pos = GetKdTreePosition(D_kd_tree, D_V.elements[query_x], D_V.elements[query_x + 1], D_V.elements[query_x +2] );
		
		float result_x = D_Result_Normals.elements[query_x ];
		float result_y = D_Result_Normals.elements[query_x + 1 ];
		float result_z = D_Result_Normals.elements[query_x + 2 ];
		
        calculateNormalFromSubtree(D_V, D_kd_tree, pos, k, result_x, result_y, result_z, method);

		D_Result_Normals.elements[query_x ] = result_x;
		D_Result_Normals.elements[query_x + 1 ] = result_y;
		D_Result_Normals.elements[query_x + 2 ] = result_z;
		
	}
	
}

/// HOST FUNCTIONS ///

void CalcNormalsCuda::init(){
	// set default k
	this->m_k = 50;
	
	// set default flippoint
	this->m_vx = 1000000.0;
	this->m_vy = 1000000.0;
	this->m_vz = 1000000.0;
	
	this->m_calc_method = 0;
}

CalcNormalsCuda::CalcNormalsCuda(PointArray& points)
{
	this->init();
	
	CalcNormalsCuda::getCudaInformation();
	
	this->V.dim = points.dim;
	
	this->V.width = points.width;
	
	mallocPointArray(V);
	
	for(int i = 0; i<points.width*points.dim; i++)
	{
		
		this->V.elements[i] = points.elements[i];
		
	}
	
	this->initKdTree();
	
}

void CalcNormalsCuda::getCudaInformation()
{
	
	m_mps = 0;
	m_threads_per_mp = 0;
	m_threads_per_block = 0;
	m_size_thread_block = new int(3);
	m_size_grid = new int(3);
	m_device_global_memory = 0;
	
	
	hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    
    
    m_mps = deviceProp.multiProcessorCount;
    m_threads_per_mp = deviceProp.maxThreadsPerMultiProcessor;
    m_threads_per_block = deviceProp.maxThreadsPerBlock;
    m_size_thread_block[0] = deviceProp.maxThreadsDim[0];
    m_size_thread_block[1] = deviceProp.maxThreadsDim[1];
    m_size_thread_block[2] = deviceProp.maxThreadsDim[2];
    m_size_grid[0] = deviceProp.maxGridSize[0];
    m_size_grid[1] = deviceProp.maxGridSize[1];
    m_size_grid[2] = deviceProp.maxGridSize[2];
    m_device_global_memory = (unsigned long long) deviceProp.totalGlobalMem;
    
}

void CalcNormalsCuda::getNormals(PointArray& output_normals)
{
	
	output_normals.dim = this->Result_Normals.dim;
	output_normals.width = this->Result_Normals.width;
	output_normals.elements = (float*)malloc( this->Result_Normals.dim * this->Result_Normals.width * sizeof(float) ) ;
	
	for(int i = 0; i< this->Result_Normals.dim * this->Result_Normals.width; i++)
	{	
		output_normals.elements[i] = this->Result_Normals.elements[i];
	}
	
}

void CalcNormalsCuda::mallocPointArray(PointArray& m) {

	m.elements = (float*)malloc(m.width * m.dim * sizeof(float));

}

void CalcNormalsCuda::generateHostPointArray(PointArray& m, int width, int dim)
{
	
	m.dim = dim;
	m.width = width;
	m.elements = (float*)malloc(m.width * m.dim * sizeof(float) );
	
}

void CalcNormalsCuda::generateDevicePointArray(PointArray& D_m, int width, int dim) {
	
    D_m.width = width;
    D_m.dim = dim;
    size_t size = D_m.width * D_m.dim * sizeof(float);
    HANDLE_ERROR( hipMalloc(&D_m.elements, size) );
    
}

void CalcNormalsCuda::copyToDevicePointArray(PointArray& m, PointArray& D_m) {
	
	size_t size = m.width * m.dim * sizeof(float);
    HANDLE_ERROR( hipMemcpy(D_m.elements, m.elements, size, hipMemcpyHostToDevice) );

}

void CalcNormalsCuda::copyToHostPointArray(PointArray& D_m, PointArray& m) {
	
	size_t size = m.width * m.dim * sizeof(float);
    HANDLE_ERROR( hipMemcpy(m.elements, D_m.elements, size, hipMemcpyDeviceToHost) );
	
}

void CalcNormalsCuda::fillPointArrayWithSequence(PointArray& m) {

    for(int i=0;i<m.width*m.dim;i++)
    {
		m.elements[i] = i;
	}

}  

void CalcNormalsCuda::copyDimensionToPointArray(PointArray& in, int dim, PointArray& out) {

    for(int i = 0; i<out.width; i++)
    {
		out.elements[i] = in.elements[i * in.dim + dim];
	}
}

void CalcNormalsCuda::copyVectorInterval(PointArray& in, int start, int end, PointArray& out) {

    for(int i=0; i < (end-start); i++)
    {
		out.elements[i] = in.elements[i + start];
	}
}

void CalcNormalsCuda::mergeHostWithIndices(float* a, float* b, int i1, int j1, int i2, int j2, int limit) {

    int limit_end = limit;
	
	float* temp = (float*) malloc((j2-i1+1) * sizeof(float));  //array used for merging
    int* temp_indices = (int*) malloc((j2-i1+1) * sizeof(int));  //array used for merging
    
    int i,j,k;
    i=i1;    //beginning of the first list
    j=i2;    //beginning of the second list
    k=0;
    
    int counter = 0;

    while( i<=j1 && j<=j2 && limit!=0 )    //while elements in both lists
    {
		counter ++;
		limit--;
        if(a[i]<a[j]){
			temp_indices[k] = b[i]; 
            temp[k++]=a[i++];
            
        }else{
			temp_indices[k] = b[j];
            temp[k++]=a[j++];
		}
    }
    
    while(i <= j1 && limit != 0) //copy remaining elements of the first list
    {
		temp_indices[k] = b[i]; 
        temp[k++]=a[i++];
	}
        
    while(j <= j2 && limit!=0 ) {   //copy remaining elements of the second list
        temp_indices[k] = b[j]; 
        temp[k++]=a[j++];
	}
        
    //Transfer elements from temp[] back to a[]
    for(i=i1,j=0;i<=j2 && limit_end!=0 ;i++,j++,limit_end--)
	{
		b[i] = temp_indices[j];
		if(b[i] < 0){
			printf("THERE IS SOMETHING WRONG\n");
		}
        a[i] = temp[j];
    }

    free(temp_indices);
    free(temp);
}


void CalcNormalsCuda::naturalMergeSort(PointArray& in, int dim, PointArray& indices, PointArray& m, int limit) {
	
	copyDimensionToPointArray(in, dim, m);

	int m_elements = m.width * m.dim;
	
	int slide_buffer_size = int(m_elements-0.5);
	int* slide_buffer = (int*) malloc(slide_buffer_size * sizeof(int));
	

	//create RUNS
	int num_slides = 1;
	slide_buffer[0] = 0;
	for(int i=1; i < slide_buffer_size+1; i++) 
	{
		if(m.elements[i] < m.elements[i-1])
		{
			slide_buffer[num_slides] = i;
			num_slides++;
		}
		
	}
	slide_buffer[num_slides] = m_elements;
	slide_buffer_size = num_slides+1;
	
	
	//sort 
	int count = 0;
	int current_limit = -1;
	while(num_slides > 1)
	{
		if(num_slides > 2){
			current_limit = limit;
		}
		
		int i;
		
		for(i=2;i<int(num_slides+1);i+=2)
		{
				
			mergeHostWithIndices(m.elements, indices.elements , slide_buffer[i-2], slide_buffer[i-1]-1, slide_buffer[i-1], slide_buffer[i]-1, current_limit);
			
			
			slide_buffer[i/2]= slide_buffer[i];
		}
		
		if(num_slides%2 == 1){
			slide_buffer[(num_slides+1)/2] = slide_buffer[num_slides];
		}
		
		count ++;
		num_slides = int(num_slides/2.0+0.5);
		
	}
	
	free(slide_buffer);
}

void CalcNormalsCuda::sortByDim(PointArray& V, int dim, PointArray& indices, PointArray& values) {

    naturalMergeSort(V, dim, indices, values);

}

void CalcNormalsCuda::splitPointArray(PointArray& I, PointArray& I_L, PointArray& I_R) {
	
	int i=0;
	for(; i < I_L.width * I_L.dim; i++){
		I_L.elements[i] = I.elements[i];
	}
	int j=0;
	for(; i<I.width*I.dim && j<I_R.width*I_R.dim; i++, j++){
		I_R.elements[j] = I.elements[i];
	}
	
}

void CalcNormalsCuda::splitPointArrayWithValue(PointArray& V, PointArray& I, PointArray& I_L, PointArray& I_R, int current_dim, float value) {

    int i_l = 0;
	int i_r = 0;
	
	for(int i=0; i<I.width; i++)
	{
		float current_value = V.elements[static_cast<int>(I.elements[i] + 0.5) * V.dim + current_dim ];
		//~ printf("curr val: %f\n", current_value);
		if(current_value <= value && I_L.width > i_l ){
			//~ printf("add to left: %f with value %f\n", I.elements[i], current_value);
			I_L.elements[i_l++] = I.elements[i];
		}else if(current_value >= value && I_R.width > i_r){
			//~ printf("add to right: %f with value %f\n", I.elements[i], current_value);
			I_R.elements[i_r++] = I.elements[i];
		}else {
			if(i_r<I_R.width){
				I_R.elements[i_r++] = I.elements[i];
			}else if(i_l<I_L.width){
				I_L.elements[i_l++] = I.elements[i];
			}
		}
	}
		
}


void CalcNormalsCuda::generateKdTreeRecursive(PointArray& V, PointArray* sorted_indices, int current_dim, int max_dim, PointArray& kd_tree, int size, int max_tree_depth, int position) {
	
	int left = position*2+1;
	int right = position*2+2;
	
    if( right > size-1 || left > size-1 )
    {

		kd_tree.elements[position] = sorted_indices[current_dim].elements[0];

    } else {
		/// split sorted_indices
		int indices_size = sorted_indices[current_dim].width;
		
        int v = pow( 2, static_cast<int>(log2f(indices_size-1) ) );
		int left_size = indices_size - v/2;

        if( left_size > v )
        {
			left_size = v;
		}
		int right_size = indices_size - left_size;
		
        float split_value = ( V.elements[current_dim+static_cast<int>(sorted_indices[current_dim].elements[left_size-1])*V.dim ] + V.elements[current_dim+static_cast<int>(sorted_indices[current_dim].elements[left_size] ) * V.dim] ) /2.0;
		
        kd_tree.elements[ position ] = split_value;
		
		struct PointArray sorted_indices_left[max_dim];
		struct PointArray sorted_indices_right[max_dim];
		
		// alloc new memory
        for( int i=0; i<max_dim; i++ )
        {
            // memory corruption when malloc
			
			sorted_indices_left[i].width = left_size;
			sorted_indices_left[i].dim = 1;
			sorted_indices_left[i].elements = (float*)malloc( (left_size+1) *sizeof(float) );
			
			sorted_indices_right[i].width = right_size;
			sorted_indices_right[i].dim = 1;
			sorted_indices_right[i].elements = (float*)malloc( (right_size+1) * sizeof(float) );
			
            if(i==current_dim){
				splitPointArray( sorted_indices[i], sorted_indices_left[i], sorted_indices_right[i]);
			}else{
				splitPointArrayWithValue(V, sorted_indices[i], sorted_indices_left[i], sorted_indices_right[i], current_dim, split_value);
			}
			
		}
		
		generateKdTreeRecursive(V, sorted_indices_left, (current_dim+1)%max_dim, max_dim, kd_tree, size, max_tree_depth, left);
		generateKdTreeRecursive(V, sorted_indices_right, (current_dim+1)%max_dim, max_dim, kd_tree, size, max_tree_depth, right);		
	
		
		// alloc new memory
        for(int i=0; i<max_dim; i++)
        {
            free( sorted_indices_left[i].elements );
            free( sorted_indices_right[i].elements );
		}
	}
	
	
}

void CalcNormalsCuda::generateKdTreeArray(PointArray& V, PointArray* sorted_indices, int max_dim, PointArray& kd_tree) {

    int size;
	int max_tree_depth;
	
	max_tree_depth = static_cast<int>( log2f(V.width - 1 ) + 2.0 ) ;

	if (V.width == 1)
	{
		max_tree_depth = 1;
	}
	
	size = V.width * 2 - 1;
	
    generateHostPointArray(kd_tree, size, 1);
	
	//start real generate
	generateKdTreeRecursive(V, sorted_indices, 0, max_dim, kd_tree, size, max_tree_depth, 0);
	
}

void CalcNormalsCuda::GPU_NN(PointArray& D_V, PointArray& D_kd_tree, PointArray& D_Result_Normals, PointArray& Result_Normals) {
	
    int threadsPerBlock = this->m_threads_per_block;
	int blocksPerGrid = (D_V.width + threadsPerBlock-1)/threadsPerBlock;

    // kNN-search and Normal calculation
    KNNKernel<<< blocksPerGrid, threadsPerBlock >>>(D_V, D_kd_tree, D_Result_Normals, this->m_k, this->m_calc_method);
    hipDeviceSynchronize();

    // Flip normals to view point
    FlipNormalsKernel<<< blocksPerGrid, threadsPerBlock >>>(D_V, D_Result_Normals, this->m_vx, this->m_vy, this->m_vz);
	hipDeviceSynchronize();

    //TODO: Interpolate
	
    size_t size = Result_Normals.width * Result_Normals.dim * sizeof(float);

    HANDLE_ERROR( hipMemcpy(Result_Normals.elements, D_Result_Normals.elements, size, hipMemcpyDeviceToHost ) );

}

void CalcNormalsCuda::initKdTree() {

	struct PointArray indices_sorted[this->V.dim];
	struct PointArray values_sorted[this->V.dim];
	
	for(int i=0; i < this->V.dim; i++)
	{
        generateHostPointArray(indices_sorted[i], V.width, 1);
		
        generateHostPointArray(values_sorted[i], V.width, 1);
		
		fillPointArrayWithSequence(indices_sorted[i]);
		
		sortByDim( this->V, i, indices_sorted[i] , values_sorted[i]);
	}

	generateKdTreeArray(V, indices_sorted, this->V.dim, this->kd_tree);
	
	for(int i=0; i<V.dim;i++)
	{
		free(indices_sorted[i].elements);
		free(values_sorted[i].elements);
	}

}

void CalcNormalsCuda::setK(int k) {

    this->m_k = k;

}

void CalcNormalsCuda::setFlippoint(float v_x, float v_y, float v_z) {

    this->m_vx = v_x;
	this->m_vy = v_y;
	this->m_vz = v_z;

}

void CalcNormalsCuda::setMethod(std::string method) {

    if( strcmp( method.c_str(), "PCA") == 0 ){
		this->m_calc_method = 0;
	} else if( strcmp( method.c_str(), "RANSAC") == 0){
		this->m_calc_method = 1;
	} else {
		printf("WARNING: Normal Calculation Method is not implemented\n");
	}

}

void CalcNormalsCuda::printSettings() {

	printf("	Nearest Neighbors = %d\n",this->m_k);
	
	printf("	Flip point = (%f, %f, %f)\n", this->m_vx, this->m_vy, this->m_vz);
	
	switch(this->m_calc_method){
		case 0:
			printf("	Method = 'PCA'\n");
			break;
		case 1:
			printf("	Method = 'RANSAC'\n");
			break;
	}
	
	printf("\n");

}

void CalcNormalsCuda::start() {
	
	generateHostPointArray( this->Result_Normals, V.width, V.dim);
	
	PointArray D_V, D_kd_tree, D_Result_Normals;
	generateDevicePointArray( D_V, this->V.width, this->V.dim );
	generateDevicePointArray( D_kd_tree, this->kd_tree.width, this->kd_tree.dim);
	generateDevicePointArray( D_Result_Normals, this->Result_Normals.width, this->Result_Normals.dim);
	
	//COPY STUFF
	copyToDevicePointArray( V, D_V);
	copyToDevicePointArray( this->kd_tree, D_kd_tree);
	
	//Cuda Kernels
	GPU_NN(D_V, D_kd_tree, D_Result_Normals, this->Result_Normals);
	
	hipFree(D_V.elements);
	hipFree(D_kd_tree.elements);
	hipFree(D_Result_Normals.elements);
		
}

CalcNormalsCuda::~CalcNormalsCuda() {

    // clearn up resulting normals and kd_tree
    // Pointcloud has to be cleaned up by user

	if(this->Result_Normals.width > 0){
		free(Result_Normals.elements);
	}
	
	if(this->kd_tree.width > 0){
		free(this->kd_tree.elements);
	}
}


