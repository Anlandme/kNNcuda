#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// Thread block size
#define BLOCK_SIZE 1

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride; 
    float* elements;
} Matrix;

void mallocMatrix(Matrix& m){
	m.elements = (float*)malloc(m.width * m.height * sizeof(float));
}

void fillMatrixWithRandomFloats(Matrix& m)
{
	int i;
	//int j;
	for(i=0;i<m.height*m.width;i++)
	{
		*(m.elements + i ) = ((float)rand()/(float)(RAND_MAX)) * 10.0 -5.0 ;
		
	}
}

void printMatrix(Matrix m)
{
	int i;
	//int j;
	for(i=0;i<m.width*m.height;i++)
	{
		if(i%m.width == 0){
			printf("|");
		}
		printf(" %f ",*(m.elements + i ));
		if(i%m.width == m.width-1){
			printf("|\n");
		}
	}
	
	printf("\n");
}

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);



// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix& A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix& A, int row, int col) 
{

    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}




// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(Matrix& A, Matrix& B, Matrix& C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);         
            
    Matrix d_B;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);

    hipMalloc(&d_B.elements, (int)size);
    hipMemcpy(d_B.elements, B.elements, size,
    hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

	clock_t calcstart, calcend;
	calcstart = clock();
	
    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

	calcend=clock();
	printf("Multiplikation %f milliseconds\n",(float)(calcend-calcstart)*1000.0 / CLOCKS_PER_SEC);
	
    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);


    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}



int main(int argc, char** argv)
{
	clock_t prgstart, prgend;
	prgstart = clock();
	
	int seed = 1479731956;
	//int seed = time(NULL);
	printf("%d\n",seed);
    srand(seed);
    
    // ergebnis matrix: firstMatrixHeight x secondMatrixWidth
    int firstMatrixHeight = 500;
    int secondMatrixWidth = 200;
    int firstSecondLength = 500;
    if(argc > 1){
		firstSecondLength = atoi(argv[1]);
		
	}
	
	printf("%dx%d * %dx%d\n",firstSecondLength,firstMatrixHeight,secondMatrixWidth,firstSecondLength);
	
	Matrix hMatA;
	hMatA.height = firstMatrixHeight;
	hMatA.width = firstSecondLength;
	hMatA.stride = hMatA.width;
	mallocMatrix(hMatA);
	fillMatrixWithRandomFloats(hMatA);
	//printMatrix(hMatA);
	
	Matrix hMatB;
	hMatB.height = firstSecondLength;
	hMatB.width = secondMatrixWidth;
	hMatB.stride = hMatB.width;
	mallocMatrix(hMatB);
	fillMatrixWithRandomFloats(hMatB);
	//printMatrix(hMatB);
	
	Matrix hMatC;
	hMatC.height = firstMatrixHeight;
	hMatC.width = secondMatrixWidth;
	hMatC.stride = hMatC.width;
	mallocMatrix(hMatC);
	
	
	printf("MULTIPLY!\n");
	MatMul(hMatA, hMatB, hMatC);
	
	
	//printMatrix(hMatC);
	
	free(hMatA.elements);
	free(hMatB.elements);
	free(hMatC.elements);
	
	prgend=clock();
	printf("Laufzeit insgesamt %f seconds\n",(float)(prgend-prgstart) / CLOCKS_PER_SEC);
	
}

